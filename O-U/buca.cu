#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define n 40 //descrivo n*n particelle
#define k 0.6 // costante nel processo di O-U
#define D 0.3
#define dt 0.1
#define durata 3000
#define termalizzazione 70

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che sta utilizza thread richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}
__global__ void inizializza(float *x){int id=threadIdx.x+ blockIdx.x*blockDim.x;
					x[id]=0;}

__global__ void evolvi(hiprandState* stato,float* x){float w;int id=threadIdx.x+ blockIdx.x*blockDim.x;	
							w=hiprand_normal(&stato[id]);	
							x[id]=x[id]-k*x[id]*dt+D*w*sqrt(dt); 
							
}
void 			stampa(float* x,int m)	{int i;for(i=0;i<m;i++) printf("%1.4f \n",x[i]);}
void		stampa_file(float*x, int m){FILE*f; f=fopen("traettoria.txt","w");int i;
					for(i=0;i<m;i++){fprintf(f,"%f\n",x[i]);}
							
					fclose(f);}

main(){float * x;float* dev_x;int N=n*n;int t;
hipEvent_t start,stop;
float *traettoria;
traettoria=(float*)malloc(durata*sizeof(float));
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
x=(float*)malloc(N*sizeof(float));
//hipMemset((float*)&dev_x,0,N*sizeof(float));
hipMalloc((float**)&dev_x,N*sizeof(float));
hipMalloc((float**)&dev_x,N*sizeof(float));
inizializza<<<n,n>>>(dev_x);
hiprandState * stato;
hipMalloc((void**)&stato,N*N*sizeof(hiprandState));
setup_random_kernel<<<n,n>>>(stato);
for(t=0;t<durata;t++){	evolvi<<<n,n>>>(stato,dev_x);
			if(t>=termalizzazione)hipMemcpy(&traettoria[t-termalizzazione],dev_x,sizeof(float),hipMemcpyDeviceToHost);}//sto dicendo che aspetto t=100 per la termalizzazione
hipMemcpy(x,dev_x,N*sizeof(float),hipMemcpyDeviceToHost);

stampa(x,N);
stampa_file(traettoria,durata-termalizzazione);
free(x);
hipFree(dev_x);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
//printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}
