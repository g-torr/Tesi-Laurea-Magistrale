#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#define tau 1. // costante nel processo di O-U
#define k 0.001	//convessitù parabola potenziale armonico
#define D 0.1
#define size 3.
#define half_size size/2.
const int termalizzazione= half_size*half_size/(2.*D);
const int blocks=600;
const int threads=1024;
const float durata=size*size/(2.*D)*4.;
const float tsalva= durata/60.;
struct configurazione{
		double* x;
		double* eta;
		};
__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che sta utilizza thread richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}
__global__ void inizializza(double *x,double *eta){int id=threadIdx.x+ blockIdx.x*blockDim.x;
					x[id]=0.75;eta[id]=0.5;}

__global__ void evolvi(hiprandState* stato,double*x, double * eta,double* forza,double dt){double w;int id=threadIdx.x+ blockIdx.x*blockDim.x;	
							w=hiprand_normal(&stato[id])*sqrt(2.0);	
							eta[id]=eta[id]-(1/tau)*eta[id]*dt+sqrt(D)*w*sqrt(dt)/tau; //Ornstein Oulenbeck
              
	forza[id]=12*k*pow(x[id],-13);          
	x[id] = x[id] +forza[id]*dt + eta[id]*dt;// processo dinamico 
	  
         
					
          if (x[id]>=half_size) x[id] = x[id]-size;
					else if(x[id]< -half_size) x[id] = x[id]+size;		      
}

void 			stampa(double* x,double dt)	{FILE*f;char indirizzo [50];int i,j;int N=blocks*threads;
						sprintf(indirizzo,"./dati_%f",dt);
						f=fopen(indirizzo,"w");	
						for(i=0;i<N;i++) fprintf(f,"%1.4f \n",x[i]);
							fclose(f);}

void		stampa_file(double*x, int m,double dt){FILE*f;char indirizzo [50];int i,j;int N=blocks*threads;
						
						sprintf(indirizzo,"forza%f",dt);
            mkdir(indirizzo,0700);
            char indirizzo2[50]; sprintf(indirizzo2,"./forza%f/forza_%d",dt,m);
						f=fopen(indirizzo2,"w");
							for(i=0;i<N;i++)fprintf(f,"%f\n",x[i]);
						fclose(f);}

main(){
  configurazione sistema; int N=blocks*threads;int t; // x è il sistema  dinamico, n è il rumore
  hipEvent_t start,stop; 
  double *traettoria; double * storage; double* forza; double* dev_forza; 
  traettoria=(double*)malloc(durata*sizeof(double));
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  storage=(double*)malloc(N*sizeof(double));
	forza=(double*)malloc(N*sizeof(double));
hipMalloc((double**)&sistema.x,N*sizeof(double));
hipMalloc((double**)&sistema.eta,N*sizeof(double));
hipMalloc((double**)&dev_forza,N*sizeof(double));

hiprandState * stato;
hipMalloc((void**)&stato,N*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(stato);

double dt;int j;dt=0.2;
	int numero_passi,passi_salvataggio,passi_termalizzazione;

for(j=0;j<10;j++){
	dt=dt/2;
  printf("%f\n",dt);
 inizializza<<<blocks,threads>>>(sistema.x,sistema.eta);
  
   numero_passi=(int)(durata/dt);
   passi_salvataggio =(int)(tsalva/dt);
  passi_termalizzazione=(int)(termalizzazione/dt);
  int i=0;
  
   for(t=0;t<numero_passi;t++){
    evolvi<<<blocks,threads>>>(stato,sistema.x,sistema.eta,dev_forza,dt);
//			if(t>=termalizzazione)hipMemcpy(&traettoria[t-termalizzazione],sistema.x,sizeof(float),hipMemcpyDeviceToHost);}
    if((t% passi_salvataggio==0)&&(t>passi_termalizzazione)){
      hipMemcpy(forza,dev_forza,N*sizeof(double),hipMemcpyDeviceToHost);
      stampa_file(forza,i,dt);i++;
    }
  }

  hipMemcpy(storage,sistema.x,N*sizeof(double),hipMemcpyDeviceToHost);
  stampa(storage,dt);
 }


free(storage);
hipFree(sistema.x);
hipFree(sistema.eta);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}
