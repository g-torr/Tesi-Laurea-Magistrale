#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#define tau 0.6 // costante nel processo di O-U
#define D 0.1
#define dt 0.1
#define durata size*size*100/(2*D)
#define termalizzazione 0
#define raggio 0.3
#define mu 1
const int blocks=40;
const int threads=1024;
const float size=10.;
const float half_size=size/2.;
struct configurazione{
		float* eta;
		float* x;
		};


void creo_cartelle()					{system( "rm -rf ./forza" );
							mkdir("forza",0700);}
__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che sta utilizza thread richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}
__global__ void inizializza(float *x,float *eta){int id=threadIdx.x+ blockIdx.x*blockDim.x;
					x[id]=0;eta[id]=0.;}

__global__ void evolvi(hiprandState* stato,float*x, float * eta,float * forza,float* v){float w;int id=threadIdx.x+ blockIdx.x*blockDim.x;	
							w=hiprand_normal(&stato[id])*sqrt(2.);	
							//w=0;
							eta[id]=eta[id]-(1/tau)*eta[id]*dt+sqrt(D)*w*sqrt(dt)/tau; 
							x[id]=x[id]+eta[id]*dt;
							float delta;forza[id]=0;
						if (x[id]+raggio>half_size){v[id]=(half_size-raggio)/dt-x[id]/dt+eta[id];
									delta=x[id]+raggio-half_size;x[id]=half_size-raggio;forza[id]=delta/(dt*mu);}
						else if (x[id]-raggio<-half_size) {v[id]=(-half_size+raggio)/dt-x[id]/dt+eta[id];
									delta=x[id]-raggio +half_size;x[id]=-half_size+raggio;forza[id]=delta/(dt*mu);} 					else v[id]=eta[id];
							
										}
void 			stampa(float* x,int m)	{int i;for(i=0;i<m;i++) printf("%1.4f %1.4f \n",x[i],x[i+m]);}
void		stampa_file(float*x, int m){FILE*f;char indirizzo [50];int i,j;int N=blocks*threads;
						
						sprintf(indirizzo,"./forza/forza_%d.txt",m);
						f=fopen(indirizzo,"w");
							for(i=0;i<N;i++)fprintf(f,"%f\n",x[i]);
						fclose(f);}
void		stampa_traettoria(float*x,float*v, int m){FILE*f; f=fopen("traettoria.txt","w");int i;
					for(i=0;i<m;i++){fprintf(f,"%f	%f\n",x[i],v[i]);}
							
					fclose(f);}

					

main(){
creo_cartelle();configurazione sistema; int N=blocks*threads;int t; // x è il sistema  dinamico, n è il rumore
hipEvent_t start,stop; 
float *traettoria; float * storage;float* forza; float* dev_forza;float* velocit;float * v; 
traettoria=(float*)malloc(durata*sizeof(float));
velocit=(float*)malloc(durata*sizeof(float));//sempre per la singola traettoria

hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
storage=(float*)malloc(2*N*sizeof(float));
forza=(float*)malloc(N*sizeof(float));

hipMalloc((float**)&sistema.x,N*sizeof(float));
hipMalloc((float**)&sistema.eta,N*sizeof(float));
hipMalloc((float**)&dev_forza,N*sizeof(float));
hipMalloc((float**)&v,N*sizeof(float));

inizializza<<<blocks,threads>>>(sistema.x,sistema.eta);
hiprandState * stato;
hipMalloc((void**)&stato,N*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(stato);
int i=0;
for(t=0;t<durata;t++){	
			evolvi<<<blocks,threads>>>(stato,sistema.x,sistema.eta,dev_forza,v);
			if(t>=termalizzazione){hipMemcpy(&traettoria[t-termalizzazione],sistema.x,sizeof(float),hipMemcpyDeviceToHost);
						hipMemcpy(&velocit[t-termalizzazione],sistema.eta,sizeof(float),hipMemcpyDeviceToHost);}
if(t%1000==999)	{hipMemcpy(forza,dev_forza,N*sizeof(float),hipMemcpyDeviceToHost);stampa_file(forza,i);i++;}
hipMemcpy(storage,sistema.x,N*sizeof(float),hipMemcpyDeviceToHost);
hipMemcpy(storage+N,v,N*sizeof(float),hipMemcpyDeviceToHost);}
stampa(storage,N);
free(storage);
stampa_traettoria(traettoria,velocit,durata-termalizzazione);
hipFree(sistema.x);
hipFree(sistema.eta);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
//printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}
