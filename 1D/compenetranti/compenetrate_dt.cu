#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#define tau 0.06 // costante nel processo di O-U
#define D 0.1
//#define durata 9000
//#define termalizzazione 70
#define raggio 0.3
#define mu 1
const int blocks=10;
const int threads=1024;
const float durata=4./(2.*D)*10.;
const float tsalva= durata/10.;
struct configurazione{
		float* eta;
		float* x;
		};
__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali curand_init (seed,sequence,offset, curandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che sta utilizza thread richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}
__global__ void inizializza(float *x,float *eta){int id=threadIdx.x+ blockIdx.x*blockDim.x;
					x[id]=0;eta[id]=0;}

__global__ void evolvi(hiprandState* stato,float*x, float * eta,float * forza,float dt){float w;int id=threadIdx.x+ blockIdx.x*blockDim.x;	
							w=hiprand_normal(&stato[id])*sqrt(2.);	
							eta[id]=eta[id]-(1/tau)*eta[id]*dt+sqrt(D)*w*sqrt(dt)/tau; 
							x[id]=x[id]+eta[id]*dt;
							float delta;forza[id]=0;
							if (x[id]+raggio>1){delta=x[id]+raggio-1;x[id]=1-raggio;forza[id]=delta/(dt*mu);}
						else if (x[id]-raggio<-1) {delta=x[id]-raggio +1;x[id]=-1+raggio;forza[id]=delta/(dt*mu);} 
							
										}
void 			stampa(float* x,float dt)	{FILE*f;char indirizzo [50];int i,j;int N=blocks*threads;
						sprintf(indirizzo,"./dati_%f",dt);
						f=fopen(indirizzo,"w");	
						for(i=0;i<N;i++) fprintf(f,"%1.4f \n",x[i]);
							fclose(f);}

void		stampa_file(float*x, int m,float dt){FILE*f;char indirizzo [50];int i,j;int N=blocks*threads;
						
						sprintf(indirizzo,"forza%f",dt);
            mkdir(indirizzo,0700);
            char indirizzo2[50]; sprintf(indirizzo2,"./forza%f/forza_%d",dt,m);
						f=fopen(indirizzo2,"w");
							for(i=0;i<N;i++)fprintf(f,"%f\n",x[i]);
						fclose(f);}
					

int main(void){
configurazione sistema; int numero_passi;
int N=blocks*threads;int t; // x è il sistema  dinamico, n è il rumore
hipEvent_t start,stop; 
float *traettoria; float * storage;float* forza; float* dev_forza; 
traettoria=(float*)malloc(durata*sizeof(float));
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
storage=(float*)malloc(N*sizeof(float));
forza=(float*)malloc(N*sizeof(float));

hipMalloc((float**)&sistema.x,N*sizeof(float));
hipMalloc((float**)&sistema.eta,N*sizeof(float));
hipMalloc((float**)&dev_forza,N*sizeof(float));
inizializza<<<blocks,threads>>>(sistema.x,sistema.eta);
hiprandState * stato;
hipMalloc((void**)&stato,N*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(stato);

double dt;int j;dt=2.;


for(j=1;j<17;j++){

  dt=dt/(2.);
  printf("%f\n",dt);
  
  inizializza<<<blocks,threads>>>(sistema.x,sistema.eta);
  
  int numero_passi=(int)(durata/dt);
  int passi_salvataggio =(int)(tsalva/dt);
  int i=0;
  
  for(t=0;t<numero_passi;t++){
    evolvi<<<blocks,threads>>>(stato,sistema.x,sistema.eta,dev_forza,dt);
//			if(t>=termalizzazione)cudaMemcpy(&traettoria[t-termalizzazione],sistema.x,sizeof(float),cudaMemcpyDeviceToHost);}
    if((t% passi_salvataggio==0)&&(t>0)){
      hipMemcpy(forza,dev_forza,N*sizeof(float),hipMemcpyDeviceToHost);
      stampa_file(forza,i,dt);i++;
    }
  }

  hipMemcpy(storage,sistema.x,N*sizeof(float),hipMemcpyDeviceToHost);
  stampa(storage,dt);
 }


free(storage);
hipFree(sistema.x);
hipFree(sistema.eta);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
//printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}
