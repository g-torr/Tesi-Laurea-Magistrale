#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#define N 4 
__global__ void somma(int* a, int* b, int* c,int *j){
int i=blockIdx.x; *j=i;
if (i<N) c[i]=a[i]+b[i];
}


int main(){
printf("è l'esercizio di pag 41 del libro");
int* a;int*b;int *c;
int *dev_a; int* dev_b; int* dev_c;
hipMalloc((void**)&dev_a, N*sizeof(int));
hipMalloc((int**)&dev_b, N*sizeof(int));
hipMalloc((void**)&dev_c,N*sizeof(int));

a=(int*)malloc(N*sizeof(int));
b=(int*)malloc(N*sizeof(int));
c=(int*)malloc(N*sizeof(int));
int i;
for(i=0;i<N;i++){
//printf("acquisisci a, b \n");
*(a+i)=i;
*(b+i)=i*i;}
hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
int j; int* dev_j;                     	//j è il  blockIdx.x che copierò dal device all'host
hipMalloc((void**)&dev_j,sizeof(int));
somma<<<N,1>>>(dev_a,dev_b,dev_c,dev_j);
hipMemcpy(c,dev_c,N*sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(&j,dev_j,sizeof(int),hipMemcpyDeviceToHost);
printf("il blockIdx.x =  %d",j);

for(i=0;i<N;i++){
printf("il tuo numero è %d \n",*(c+i));}

return 0;}
