#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define N 20*50000
#define imin(a,b) (a<b?a:b)
const int threadsperblock= 256;
const int numberofblocks=imin(32,(N+ threadsperblock-1)/threadsperblock);

__global__ void somma(float *a, float *somme){int i= threadIdx.x+ blockIdx.x *blockDim.x;
	__shared__ float parziale[threadsperblock]; //si sfrutta la proprietà della shared memory, infatti vengono creati numberofblocks copie della variabile  parziale, cosicchè l'indice di parziale gira solo sui treads di un singolo blocco, 
		int cacheindex=threadIdx.x;
	float temp=0;
	while(i<N){temp=temp + *(a+i);
			i=i+ blockDim.x*gridDim.x;}
	parziale[cacheindex]=temp; // in questo passaggio l'indice di parziale gira solo sui treads di un singolo blocco, ma concretamente noi abbiamo   numberofblocks  di questi threads che mi ricoprono tutti gli N elementi (in realtà l'ultimo blocco potrebbe rimanere incompleto se N non è un multiplo di threadsperblock)

	__syncthreads();
	int k=blockDim.x/2;  //riduzione, funziona solo per un threadsperblock pari a una potenza di 2
	while(k!=0){
	if(cacheindex<k) parziale[cacheindex]+=parziale[cacheindex+k];
			__syncthreads();	k=k/2;}
	if (cacheindex==0) somme[blockIdx.x] =parziale[0];
						}


void stampa(float *a,int n){int i;
		for(i=0;i<n;i++)printf("%f\n",a[i]);}

__global__ void somma2(float *a,float media,float* somme2){
			int i=threadIdx.x+blockIdx.x *blockDim.x ;
			float temp=0;
			while(i<N){	temp=temp+a[i]*a[i];	 i+=blockDim.x*gridDim.x;}
		__shared__ float parziale[threadsperblock];
			parziale[threadIdx.x]=temp;
			__syncthreads();
			int j= blockDim.x/2;
			while (j!=0){
					if(threadIdx.x< j) parziale[threadIdx.x]+= parziale[threadIdx.x+j];
					__syncthreads(); j=j/2; }
			if(threadIdx.x==0)	somme2[blockIdx.x]= parziale[0];
						}

main(){ float* a; float* dev_a;
float *somme; float* dev_somme;
int i;
srand(time(NULL));
a=(float*)malloc(N*sizeof(float));
somme=(float*)malloc(numberofblocks*sizeof(float));
hipMalloc((void**)&dev_somme,numberofblocks*sizeof(float));
hipMalloc((void**)&dev_a, N*sizeof(float));
for(i=0;i<N;i++) *(a+i)=(float)rand()/RAND_MAX;
hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(dev_somme,somme,numberofblocks*sizeof(float),hipMemcpyHostToDevice);
somma<<<numberofblocks,threadsperblock>>>(dev_a,dev_somme);
hipMemcpy(somme,dev_somme,numberofblocks*sizeof(float),hipMemcpyDeviceToHost);
//stampa(somme,numberofblocks);
float media=0;
for(i=0;i<numberofblocks;i++){
media=media+ *(somme+i);
}
int n=N;
media= media /n;
printf("la media è %f \n",media);

// ora calcolo la varianza
float * somme2; float * dev_somme2;
somme2=(float*)malloc(numberofblocks*sizeof(float));
hipMalloc((void**)&dev_somme2,numberofblocks*sizeof(float));
somma2<<<numberofblocks,threadsperblock>>>(dev_a,media,dev_somme2);
hipMemcpy(somme2,dev_somme2,numberofblocks*sizeof(float),hipMemcpyDeviceToHost);
float varianza=0;
stampa (somme2,numberofblocks);
for(i=0;i<numberofblocks;i++){varianza+=somme2[i];}		
			varianza=varianza/n- media*media;
	printf("varianza = %f\n",varianza);
return 0;
}
