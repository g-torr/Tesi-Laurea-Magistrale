#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

/*__global__ void somma(float *a,float *b, float *c, int N){
		int i=threadIdx.x+ blockIdx.x * blockDim.x;
			while(i<N){*(c+i)=*(a+i)+*(b+i);
				i=i+(gridDim.x*blockDim.x);}}*/
main(){ 
clock_t t1 = clock();
srand(time(NULL));
	int N,i;
printf("inserire il numero di elementi del vettore");
scanf("%d",&N);
float *a; float *b; float *c;

a=(float*)malloc(N*sizeof(float));
b=(float*)malloc(N*sizeof(float));
c=(float*)malloc(N*sizeof(float));
for(i=0;i<N;i++){*(a+i)=(float)rand()/RAND_MAX;
		*(b+i)=(float)rand()/RAND_MAX;}
float *dev_a; float * dev_b; float *dev_c;
/*hipMalloc((void**)&dev_a, N*sizeof(float));
hipMalloc((void**)&dev_b, N*sizeof(float));
hipMalloc((void**)&dev_c, N*sizeof(float));
hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice);
somma<<<12,12>>>(dev_a,dev_b,dev_c,N);
hipMemcpy(c,dev_c,N*sizeof(float),hipMemcpyDeviceToHost);
*/
for(i=0;i<N;i++){*(c+i)=*(a+i)+*(b+i);}

//for(i=0;i<N;i++){
//printf("%f + %f = %f \n",*(a+i),*(b+i),*(c+i));}

 clock_t t2 = clock();
double time_sec = 
       (double)(t2-t1)/(double)(CLOCKS_PER_SEC); 
 
    printf("Time (sec): %lf\n",time_sec); 
return 0;
}

