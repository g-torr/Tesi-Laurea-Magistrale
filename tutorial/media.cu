#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define N 20000*5000
#define imin(a,b) (a<b?a:b)
const int threadsperblock= 256;
const int numberofblocks=imin(32,(N+ threadsperblock-1)/threadsperblock);

//tramite la shared memory sto praticamente facendo la somma in blocchi

__global__ void somma(float *a, float *somme){int i= threadIdx.x+ blockIdx.x *blockDim.x;
	__shared__ float parziale[threadsperblock]; //si sfrutta la proprietà della shared memory, infatti vengono creati numberofblocks  di diverse variabili  parziale, cosicchè l'indice di parziale gira solo sui treads di un singolo blocco, inoltre i threads di ogni variabile shared possono comunicare
		int cacheindex=threadIdx.x;
	float temp=0;
	while(i<N){temp=temp + *(a+i);
			i=i+ blockDim.x*gridDim.x;}
	parziale[cacheindex]=temp; // in questo passaggio l'indice di parziale gira solo sui treads di un singolo blocco, ma concretamente noi abbiamo   numberofblocks  di questi threads che mi ricoprono tutti gli N elementi (in realtà l'ultimo blocco potrebbe rimanere incompleto se N non è un multiplo di threadsperblock)

__syncthreads();
int k=blockDim.x/2;  //riduzione, funziona solo per un threadsperblock = 2^n, qui sfrutto la possibilità  di condividere la memoria
while(k!=0){
	if(cacheindex<k) parziale[cacheindex]+=parziale[cacheindex+k];
			__syncthreads();	k=k/2;}
if (cacheindex==0) somme[blockIdx.x] =parziale[0];
}


void stampa(float *a,int n){int i;
for(i=0;i<n;i++)printf("%f\n",a[i]);}

main(){ float* a; float* dev_a;
float *somme; float* dev_somme;
int i;
srand(4);
a=(float*)malloc(N*sizeof(float));
somme=(float*)malloc(numberofblocks*sizeof(float));
hipMalloc((void**)&dev_somme,numberofblocks*sizeof(float));
hipMalloc((void**)&dev_a, N*sizeof(float));
for(i=0;i<N;i++) *(a+i)=(float)rand()/RAND_MAX;
hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(dev_somme,somme,numberofblocks*sizeof(float),hipMemcpyHostToDevice);
somma<<<numberofblocks,threadsperblock>>>(dev_a,dev_somme);
hipMemcpy(somme,dev_somme,numberofblocks*sizeof(float),hipMemcpyDeviceToHost);
stampa(somme,numberofblocks);
float media=0;
for(i=0;i<numberofblocks;i++){
media=media+ *(somme+i);
//printf("la media è %f \n",media);
}
int n=N;
media= media /n;
printf("la media è %f \n",media);

return 0;
}
