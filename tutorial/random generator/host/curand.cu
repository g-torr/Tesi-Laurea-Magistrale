/*
 * This program uses the host CURAND API to generate 100 
 * pseudorandom floats.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

void stampa(float* x,int n){int i; for(i = 0; i < n; i++) {
        printf("%1.4f \n", x[i]);
    }
    printf("\n");}
void crea_hist(float*x, int n,float *hist,float h,float k,float a){int i,j,l; 
						

                                                

                                                

 

                                           for(i=0;i<h;i++){//fa aumentare la classe di freq

                                           l=0;*(hist+i)=0;

                                           

                                                       for (j=0;j<n;j++){ // fa girare la x

                                                   

                                                                             if((*(x+j)<(a+((i+1)*k)))&&(*(x+j)>(a+(i*k))))

                                                                             {l=l+1;

										*(hist+i)=l; 

                                                                                                                     } } 

                                                                                                                   }
									stampa(hist,h);}           

                            
int main(int argc, char *argv[])
{
    size_t n = 2000;
    size_t i;
    hiprandGenerator_t gen;
    float *devData, *hostData;

    /* Allocate n floats on host */
    hostData = (float *)calloc(n, sizeof(float));

    /* Allocate n floats on device */
   hipMalloc((void **)&devData, n*sizeof(float));

    /* Create pseudo-random number generator */
   hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);//Ho letto però che nelle Monte Carlo in più dimensioni è conveniente usare quasirandom
//    curandCreateGenerator(&gen,CURAND_RNG_QUASI_SCRAMBLED_SOBOL32); // è un quasirandon famiglia SOBOL 32 scrabled
    /* Set seed */
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    /* Generate n floats on device */
  hiprandGenerateUniform(gen, devData, n);
/*	float mean=0; float stddev=1;
    curandGenerateNormal(gen,devData,n,mean,stddev);
*/	

    /* Copy device memory to host */
    hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);

    /* Show result */
  stampa(hostData,n);

/*	float* hist; int h=20;hist=(float*)malloc(h*sizeof(float));
	float k=(float)6/h;
	crea_hist(hostData,n,hist,h,k,-3);*/
	
    /* Cleanup */
    hiprandDestroyGenerator(gen);
    hipFree(devData);
    free(hostData);    
    return EXIT_SUCCESS;
}





