#include "hip/hip_runtime.h"
/*
 * This program uses the device CURAND API to calculate what 
 * proportion of pseudo-random ints have low bit set.
 * It then generates uniform results to calculate how many
 * are greater than .5.
 * It then generates  normal results to calculate how many 
 * are within one standard deviation of the mean.
 */
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void setup_kernel(hiprandStatePhilox4_32_10_t *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void setup_kernel(hiprandStateMRG32k3a *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(0, id, 0, &state[id]);
}

__global__ void generate_kernel(hiprandState *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&localState);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_kernel(hiprandStatePhilox4_32_10_t *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandStatePhilox4_32_10_t localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&localState);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_uniform_kernel(hiprandState *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int i = 0; i < n; i++) {
        x = hiprand_uniform(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_uniform_kernel(hiprandStatePhilox4_32_10_t *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float x;
    /* Copy state to local memory for efficiency */
    hiprandStatePhilox4_32_10_t localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int i = 0; i < n; i++) {
        x = hiprand_uniform(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_normal_kernel(hiprandState *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random normals */
    for(int i = 0; i < n/2; i++) {
        x = hiprand_normal2(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_normal_kernel(hiprandStatePhilox4_32_10_t *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandStatePhilox4_32_10_t localState = state[id];
    /* Generate pseudo-random normals */
    for(int i = 0; i < n/2; i++) {
        x = hiprand_normal2(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_kernel(hiprandStateMRG32k3a *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    unsigned int x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&localState);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_uniform_kernel(hiprandStateMRG32k3a *state,
                                int n, 
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    double x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random uniforms */
    for(int i = 0; i < n; i++) {
        x = hiprand_uniform_double(&localState);
        /* Check if > .5 */
        if(x > .5) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

__global__ void generate_normal_kernel(hiprandStateMRG32k3a *state, 
                                int n,
                                unsigned int *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    unsigned int count = 0;
    double2 x;
    /* Copy state to local memory for efficiency */
    hiprandStateMRG32k3a localState = state[id];
    /* Generate pseudo-random normals */
    for(int i = 0; i < n/2; i++) {
        x = hiprand_normal2_double(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{

    int i;
    unsigned int total;
    hiprandState *devStates;
    hiprandStateMRG32k3a *devMRGStates;
    hiprandStatePhilox4_32_10_t *devPHILOXStates;
    unsigned int *devResults, *hostResults;
    bool useMRG = 0;
    bool usePHILOX = 0;
    int sampleCount = 10000;
    bool doubleSupported = 0;
    int device;
    struct hipDeviceProp_t properties;  

    /* check for double precision support */
    CUDA_CALL(hipGetDevice(&device));
    CUDA_CALL(hipGetDeviceProperties(&properties,device));
    if ( properties.major >= 2 || (properties.major == 1 && properties.minor >= 3) ) {
        doubleSupported = 1;
    }
    
    /* Check for MRG32k3a option (default is XORWOW) */
    if (argc >= 2)  {
        if (strcmp(argv[1],"-m") == 0) {
            useMRG = 1;
            if (!doubleSupported){
                printf("MRG32k3a requires double precision\n");
                printf("^^^^ test WAIVED due to lack of double precision\n");
                return EXIT_SUCCESS;
            }
        }else if (strcmp(argv[1],"-p") == 0) {
		usePHILOX = 1;
	} 
        /* Allow over-ride of sample count */    
        sscanf(argv[argc-1],"%d",&sampleCount); 
    }

    /* Allocate space for results on host */
    hostResults = (unsigned int *)calloc(64 * 64, sizeof(int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 64 * 
              sizeof(unsigned int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
              sizeof(unsigned int)));

    /* Allocate space for prng states on device */
    if (useMRG) {
        CUDA_CALL(hipMalloc((void **)&devMRGStates, 64 * 64 * 
                  sizeof(hiprandStateMRG32k3a)));
    }else if(usePHILOX) {
        CUDA_CALL(hipMalloc((void **)&devPHILOXStates, 64 * 64 * 
                  sizeof(hiprandStatePhilox4_32_10_t)));
    }else {
        CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 * 
                  sizeof(hiprandState)));
    }
    
    /* Setup prng states */
    if (useMRG) {
        setup_kernel<<<64, 64>>>(devMRGStates);
    }else if(usePHILOX)
    {
        setup_kernel<<<64, 64>>>(devPHILOXStates);
    }else {
        setup_kernel<<<64, 64>>>(devStates);
    }
    
    /* Generate and use pseudo-random  */
    for(i = 0; i < 50; i++) {
        if (useMRG) {
            generate_kernel<<<64, 64>>>(devMRGStates, sampleCount, devResults);
        }else if (usePHILOX){
            generate_kernel<<<64, 64>>>(devPHILOXStates, sampleCount, devResults);
	}else {
            generate_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
        }
    }
    
    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    printf("Fraction with low bit set was %10.13f\n", 
        (float)total / (64.0f * 64.0f * sampleCount * 50.0f));
        
    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
              sizeof(unsigned int)));

    /* Generate and use uniform pseudo-random  */
    for(i = 0; i < 50; i++) {
        if (useMRG) {
            generate_uniform_kernel<<<64, 64>>>(devMRGStates, sampleCount, devResults);
        }else if(usePHILOX) {
            generate_uniform_kernel<<<64, 64>>>(devPHILOXStates, sampleCount, devResults);
	}else {
            generate_uniform_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
        }
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    printf("Fraction of uniforms > 0.5 was %10.13f\n", 
        (float)total / (64.0f * 64.0f * sampleCount * 50.0f));
    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
              sizeof(unsigned int)));

    /* Generate and use normal pseudo-random  */
    for(i = 0; i < 50; i++) {
        if (useMRG) {
            generate_normal_kernel<<<64, 64>>>(devMRGStates, sampleCount, devResults);
        }else if(usePHILOX) {
            generate_normal_kernel<<<64, 64>>>(devPHILOXStates, sampleCount, devResults);
	}else {
            generate_normal_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
        }
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
        sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
    }
    printf("Fraction of normals within 1 standard deviation was %10.13f\n", 
        (float)total / (64.0f * 64.0f * sampleCount * 50.0f));

    /* Cleanup */
    if (useMRG) {
        CUDA_CALL(hipFree(devMRGStates));
    }else if(usePHILOX)
    {
        CUDA_CALL(hipFree(devPHILOXStates));
    }else {
        CUDA_CALL(hipFree(devStates));
    }    
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_example PASSED\n");
    return EXIT_SUCCESS;
}

The following example uses the cuRAND host MTGP setup API, and the cuRAND device API, to generate integers using the MTGP32 generator, and calculates the proportion that have the low bit set.

/*
 * This program uses the device CURAND API to calculate what 
 * proportion of pseudo-random ints have low bit set.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
/* include MTGP host helper functions */
#include <hiprand/hiprand_mtgp32_host.h>
/* include MTGP pre-computed parameter sets */
#include <rocrand/rocrand_mtgp32_11213.h>


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void generate_kernel(hiprandStateMtgp32 *state, 
                                int n,
                                int *result)
{
    int id = threadIdx.x + blockIdx.x * 256;
    int count = 0;
    unsigned int x;
    /* Generate pseudo-random unsigned ints */
    for(int i = 0; i < n; i++) {
        x = hiprand(&state[blockIdx.x]);
        /* Check if low bit set */
        if(x & 1) {
            count++;
        }
    }
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{
    int i;
    long long total;
    hiprandStateMtgp32 *devMTGPStates;
    mtgp32_kernel_params *devKernelParams;
    int *devResults, *hostResults;
    int sampleCount = 10000;
    
    /* Allow over-ride of sample count */    
    if (argc == 2) {
        sscanf(argv[1],"%d",&sampleCount);
    }
        
    /* Allocate space for results on host */
    hostResults = (int *)calloc(64 * 256, sizeof(int));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 256 * 
              sizeof(int)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 256 * 
              sizeof(int)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devMTGPStates, 64 * 
              sizeof(hiprandStateMtgp32)));
    
    /* Setup MTGP prng states */
    
    /* Allocate space for MTGP kernel parameters */
    CUDA_CALL(hipMalloc((void**)&devKernelParams, sizeof(mtgp32_kernel_params)));
    
    /* Reformat from predefined parameter sets to kernel format, */
    /* and copy kernel parameters to device memory               */
    CURAND_CALL(hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, devKernelParams));
    
    /* Initialize one state per thread block */
    CURAND_CALL(hiprandMakeMTGP32KernelState(devMTGPStates, 
                mtgp32dc_params_fast_11213, devKernelParams, 64, 1234));
    
    /* State setup is complete */
    
    /* Generate and use pseudo-random  */
    for(i = 0; i < 10; i++) {
        generate_kernel<<<64, 256>>>(devMTGPStates, sampleCount, devResults);
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 256 * 
        sizeof(int), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 256; i++) {
        total += hostResults[i];
    }
    
    
    printf("Fraction with low bit set was %10.13g\n", 
        (double)total / (64.0f * 256.0f * sampleCount * 10.0f));

    /* Cleanup */
    CUDA_CALL(hipFree(devMTGPStates));
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_mtgp_example PASSED\n");
    return EXIT_SUCCESS;
}


Read more at: http://docs.nvidia.com/cuda/hiprand/index.html#ixzz4MIRQ90YK
Follow us: @GPUComputing on Twitter | NVIDIA on Facebook

