#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define n 1024

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che sta utilizza thread richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			
			
				}

__global__ void fissa_lo_stato(unsigned int*states){
			unsigned int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			states[id]=id;}			
__device__ float rng_uni(unsigned int *state)
{
        // generates uniform ran num, 
        // keeps state val pointed by *state updated 

        unsigned int x = *state;

        x = x ^ (x >> 13);
        x = x ^ (x << 17);
        x = x ^ (x >> 5);

        *state = x;

        return (float) x / 4294967296;
}

__global__ void dyn_kernel(unsigned int *states,float*x)
{
	
        int id = threadIdx.x+blockIdx.x*blockDim.x;
	x[id]=rng_uni(&states[id]);
}

__global__ void genera_random(hiprandState* stato,float *x){int id=threadIdx.x+ blockIdx.x*blockDim.x;
								
							x[id]=hiprand(&stato[id]);}
void stampa(float* x,int N){int i=2;
//for(i=0;i<N;i++)
 printf("%1.4f \n",x[i]);}

main(){
float *x;float*dev_x;
int t=100;
x=(float*)malloc(n*n*sizeof(float));
hipMalloc((float**)&dev_x,n*n*sizeof(float));
unsigned int* states;
hipMalloc((unsigned int**)&states,n*n*sizeof(unsigned int));
fissa_lo_stato<<<n,n>>>(states);
//ora alloco la variabile di tipo hiprandState
hiprandState * stato;
hipMalloc((void**)&stato,n*n*sizeof(hiprandState));
setup_random_kernel<<<n,n>>>(stato);
for(int j=0;j<t;j++){
genera_random<<<n,n>>>(stato,dev_x);
dyn_kernel<<<n,n>>>(states,dev_x);
hipMemcpy(x,dev_x,n*n*sizeof(float),hipMemcpyDeviceToHost);
stampa(x,n*n);}
hipFree(dev_x);
hipFree(stato);
free(x);}
