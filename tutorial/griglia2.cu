#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define imin(a,b) (a<b?a:b)
const int DIM=4;
const int threadsperblock=2;
const int durata=10;
#define rate 0.03f

texture <float,1> text_Told;
texture <float,1> text_Tupd;
texture <float,1> text_Tfix;
struct dati{
		float* old;
		float* upd;
		};


void	stampa(float*T, int n){int i,j;for(i=0;i<n;i++){for(j=0;j<n;j++){printf("%f	",T[i+(j*n)]);}
							printf("\n");}}
	
void	stampa_file(float*T, int n){FILE*f; f=fopen("output2.txt","w");int i,j;
							fprintf(f," Vi sono n+1 matrici, la matrice 0 è la condizione iniziale, mentre l'elemento 0 del vettore di matrici updated rappresenta il primo timestep\n copio la matrice %d\n",0);
					for(i=0;i<n;i++){for(j=0;j<n;j++){fprintf(f,"%f	",T[i+(j*n)]);}
							fprintf(f,"\n");}
					fclose(f);}
void	stampa_file2(float ** storage){FILE*f; f=fopen("output2.txt","a");int i,n,j;
for(n=0;n<durata;n++){
fprintf(f," copio la matrice %d\n",n+1);
					for(i=0;i<DIM;i++){for(j=0;j<DIM;j++){fprintf(f,"%f	",storage[n][i+(j*DIM)]);}
							fprintf(f,"\n");}}
fclose(f);}


void	 constrains(float* T){	int i;
				 float p;
				for(i=0;i<(DIM*DIM);i++){p=(float)rand()/RAND_MAX;
									if(p>0.7) T[i]=1;
									else T[i]=0;	
					}
				}


void     inizializza(float* T_fix,float *T)	{ int i;
                                 float p;
                                for(i=0;i<(DIM*DIM);i++)	{if(T_fix[i]==0)	{p=(float)rand()/RAND_MAX;T[i]=p;}
                                                                        	else T[i]=1;
                                        		}			}
                               		    	 

__global__ void apply_constrains(float *old){int x,y,offset;
						 x=threadIdx.x+blockIdx.x*blockDim.x;
						 y=threadIdx.y+blockIdx.y*blockDim.y;
						offset= x+y*DIM;float c=tex1Dfetch(text_Tfix,offset);
						if(c!=0){old[offset]=1;} 
							}
							
__global__ void execute(float* upd, bool flag){int x,y,offset;
							x=threadIdx.x+blockIdx.x*blockDim.x;
							y=threadIdx.y+blockIdx.y*blockDim.y;
							offset=x+y*DIM;
							int u,b,l,r;//interazione a primi vicini:up,bottom,left,right
							//periodic boundary condition
							if(x==0) 	{u=offset+(DIM-1);b=offset+1;}
							else if(x==DIM-1){b=offset-(DIM-1);u=offset-1;}
							else		{u=offset-1;b=offset+1;}
							if (y==0)	{l=offset+(DIM-1)*DIM;r=offset+DIM;}
							else if(y==DIM-1){r=offset-(DIM-1)*DIM;l=offset-DIM;}
							else		{r=offset+DIM;l=offset-DIM;}
							float top,bottom, right,left,center;
						if (flag){	top=tex1Dfetch(text_Told,u);
								bottom=tex1Dfetch(text_Told,b);
								right=tex1Dfetch(text_Told,r);
								left=tex1Dfetch(text_Told,l);
								center=tex1Dfetch(text_Told,offset);}
						else{		top=tex1Dfetch(text_Tupd,u);
								bottom=tex1Dfetch(text_Tupd,b);
								right=tex1Dfetch(text_Tupd,r);
								left=tex1Dfetch(text_Tupd,l);
								center=tex1Dfetch(text_Tupd,offset); }			
				

						upd[offset]=center+rate*(bottom+left+right+top-4*center);

								}


void	time_step(dati* T,bool* flag)	{
			//	for(n=0;n<durata;n++){
				dim3 blocks(DIM/threadsperblock,DIM/threadsperblock);
				dim3 threads (threadsperblock,threadsperblock);
				
				float * in; float *out;
				if (*flag){in=T->old;		 out=T->upd; }
				else	{in=T->upd;		 out=T->old;}
				apply_constrains<<<blocks,threads>>>(in);
				execute<<<blocks,threads>>>(out,*flag);
					*flag=!(*flag);			
			}
				




main(){srand(5);
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
float* T_fissata;float* T_iniziale;
T_fissata=(float*)malloc(DIM*DIM*sizeof(float));
constrains(T_fissata);	//pongo i vincoli su alcune celle scelte casualmente con probabilità = 0.3

T_iniziale=(float*)malloc(DIM*DIM*sizeof(float));
inizializza(T_fissata,T_iniziale);
dati T_dev;
//T_dev=(dati*)malloc(durata*sizeof(dati));


int n=0;			//definisco matrici nella GPU

hipMalloc((void**)&T_dev.old,DIM*DIM*sizeof(float));//}
hipBindTexture(NULL,text_Told,T_dev.old,DIM*DIM*sizeof(float));	
hipMemcpy(T_dev.old,T_iniziale,DIM*DIM*sizeof(float),hipMemcpyHostToDevice);
float *dev_fissata;

hipMalloc((void**)&dev_fissata,DIM*DIM*sizeof(float));
hipBindTexture(NULL,text_Tfix,dev_fissata,DIM*DIM*sizeof(float));
hipMemcpy(dev_fissata,T_fissata,DIM*DIM*sizeof(float),hipMemcpyHostToDevice);

hipMalloc((void**)&T_dev.upd,DIM*DIM*sizeof(float));
hipBindTexture(NULL,text_Tupd,T_dev.upd,DIM*DIM*sizeof(float));

stampa_file (T_iniziale,DIM); 

float ** storage;
storage=(float**)malloc(durata*sizeof(float*));

bool  flag=true;
for(n=0;n<durata;n++){
time_step(&T_dev,&flag);
storage[n]=(float*)malloc(DIM*DIM*sizeof(float));
//hipMemcpy(storage[n],T_dev.upd,DIM*DIM*sizeof(float),hipMemcpyDeviceToHost);

if (flag)hipMemcpy(storage[n],T_dev.old,DIM*DIM*sizeof(float),hipMemcpyDeviceToHost);
else	hipMemcpy(storage[n],T_dev.upd,DIM*DIM*sizeof(float),hipMemcpyDeviceToHost);
			}
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
stampa_file2(storage);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);



/*				serviva a controllaregli indici della griglia
int indici[DIM*DIM];int j,offset;printf("stampo gli indici \n");
for(i=0;i<DIM;i++){for(j=0;j<DIM;j++){offset=i+(j*DIM); printf("%d	",offset);}
		printf("\n");}
*/
hipFree(T_dev.upd);
hipFree(dev_fissata);
hipFree(T_dev.old);



return 0;
}
