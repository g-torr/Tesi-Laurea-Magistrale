#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define imin(a,b) (a<b?a:b)
const int DIM=4;
const int threadsperblock=2;
const int durata=10;
#define rate 0.03f
/*
texture <float,2> text_Told;
texture <float,2> text_Tnew;
texture <float,2> text_Tfix;*/
struct dati{
		float* old;
		float* upd;
		};


void	stampa(float*T, int n){int i,j;for(i=0;i<n;i++){for(j=0;j<n;j++){printf("%f	",T[i+(j*n)]);}
							printf("\n");}}
	
void	stampa_file(float*T, int n){FILE*f; f=fopen("output.txt","w");int i,j;
							fprintf(f," Vi sono n+1 matrici, la matrice 0 è la condizione iniziale, mentre l'elemento 0 del vettore di matrici updated rappresenta il primo timestep\n copio la matrice %d\n",0);
					for(i=0;i<n;i++){for(j=0;j<n;j++){fprintf(f,"%f	",T[i+(j*n)]);}
							fprintf(f,"\n");}
					fclose(f);}
void	stampa_file2(float ** storage){FILE*f; f=fopen("output.txt","a");int i,n,j;
for(n=0;n<durata;n++){
fprintf(f," copio la matrice %d\n",n+1);
					for(i=0;i<DIM;i++){for(j=0;j<DIM;j++){fprintf(f,"%f	",storage[n][i+(j*DIM)]);}
							fprintf(f,"\n");}}
fclose(f);}


void	 constrains(float* T){	int i;
				 float p;
				for(i=0;i<(DIM*DIM);i++){p=(float)rand()/RAND_MAX;
									if(p>0.7) T[i]=1;
									else T[i]=0;	
					}
				}


void     inizializza(float* T_fix,float *T)	{ int i;
                                float p;
                                for(i=0;i<(DIM*DIM);i++)	{if(T_fix[i]==0)	{p=(float)rand()/RAND_MAX;T[i]=p;}
                                                                        	else T[i]=1;
                                        		}			}
                               		    	 

__global__ void apply_constrains(float *fix, float *old){int x,y,offset;
						 x=threadIdx.x+blockIdx.x*blockDim.x;
						 y=threadIdx.y+blockIdx.y*blockDim.y;
						offset= x+y*DIM;
						if(fix[offset]!=0){old[offset]=1;} 
							}
							
__global__ void execute(float* old, float* upd){int x,y,offset;
							x=threadIdx.x+blockIdx.x*blockDim.x;
							y=threadIdx.y+blockIdx.y*blockDim.y;
							offset=x+y*DIM;
							int u,b,l,r;//interazione a primi vicini:up,bottom,left,right
							//periodic boundary condition
							if(x==0) 	{u=offset+(DIM-1);b=offset+1;}
							else if(x==DIM-1){b=offset-(DIM-1);u=offset-1;}
							else		{u=offset-1;b=offset+1;}
							if (y==0)	{l=offset+(DIM-1)*DIM;r=offset+DIM;}
							else if(y==DIM-1){r=offset-(DIM-1)*DIM;l=offset-DIM;}
							else		{r=offset+DIM;l=offset-DIM;}
							upd[offset]=old[offset]+(rate*(old[u]+old[b]+old[l]+old[r]-(4*old[offset])));


								}


void	time_step(dati* T,float * T_fissata)	{
			//	for(n=0;n<durata;n++){
				dim3 blocks(DIM/threadsperblock,DIM/threadsperblock);
				dim3 threads (threadsperblock,threadsperblock);
					
				apply_constrains<<<blocks,threads>>>(T_fissata,T->old);
				execute<<<blocks,threads>>>(T->old,T->upd);
				T->old=T->upd;				
			}
				




main(){srand(5);
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
float* T_fissata;float* T_iniziale;
T_fissata=(float*)malloc(DIM*DIM*sizeof(float));
constrains(T_fissata);	//pongo i vincoli su alcune celle scelte casualmente con probabilità = 0.3

T_iniziale=(float*)malloc(DIM*DIM*sizeof(float));
inizializza(T_fissata,T_iniziale);
dati T_dev;
//T_dev=(dati*)malloc(durata*sizeof(dati));


int n=0;			//definisco matrici nella GPU

hipMalloc((void**)&T_dev.old,DIM*DIM*sizeof(float));//}
	
hipMemcpy(T_dev.old,T_iniziale,DIM*DIM*sizeof(float),hipMemcpyHostToDevice);
float *dev_fissata;

hipMalloc((void**)&dev_fissata,DIM*DIM*sizeof(float));
hipMemcpy(dev_fissata,T_fissata,DIM*DIM*sizeof(float),hipMemcpyHostToDevice);

hipMalloc((void**)&T_dev.upd,DIM*DIM*sizeof(float));
stampa_file (T_iniziale,DIM); 
float ** storage;
storage=(float**)malloc(durata*sizeof(float*));


for(n=0;n<durata;n++){
time_step(&T_dev,dev_fissata);
//hipMemcpy(T_dev.old,T_dev.upd,DIM*DIM*sizeof(float),hipMemcpyDeviceToDevice);
storage[n]=(float*)malloc(DIM*DIM*sizeof(float));
hipMemcpy(storage[n],T_dev.upd,DIM*DIM*sizeof(float),hipMemcpyDeviceToHost);
			}
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
stampa_file2(storage);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);


/*				serviva a controllaregli indici della griglia
int indici[DIM*DIM];int j,offset;printf("stampo gli indici \n");
for(i=0;i<DIM;i++){for(j=0;j<DIM;j++){offset=i+(j*DIM); printf("%d	",offset);}
		printf("\n");}
*/
hipFree(T_dev.upd);
hipFree(dev_fissata);
hipFree(T_dev.old);



return 0;
}
