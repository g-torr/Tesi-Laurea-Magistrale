#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#define tau 0.6 // costante nel processo di O-U
#define k 2.0	//convessitù parabola potenziale armonico
#define D 0.1
#define dt 0.001
#define durata 3000
#define termalizzazione 70

const int blocks=40;
const int threads=1024;
struct configurazione{
		float* x;
		float* eta;
		};
__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che sta utilizza thread richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}
__global__ void inizializza(float *x,float *eta){int id=threadIdx.x+ blockIdx.x*blockDim.x;
					x[id]=0;eta[id]=0;}

__global__ void evolvi(hiprandState* stato,float*x, float * eta){float w;int id=threadIdx.x+ blockIdx.x*blockDim.x;	
							w=hiprand_normal(&stato[id])*sqrt(2.);	
							x[id]=x[id]-(1/tau)*x[id]*dt+sqrt(D)*w*sqrt(dt)/tau; 
							eta[id]=eta[id]-k*eta[id]*dt+x[id]*dt;
							
}
void 			stampa(float* x,int m)	{int i;for(i=0;i<m;i++) printf("%1.4f \n",x[i]);}
void		stampa_file(float*x, int m){FILE*f; f=fopen("traettoria.txt","w");int i;
					for(i=0;i<m;i++){fprintf(f,"%f\n",x[i]);}
							
					fclose(f);}

main(){configurazione sistema; int N=blocks*threads;int t; // x è il sistema  dinamico, n è il rumore
hipEvent_t start,stop; 
float *traettoria; float * storage; 
traettoria=(float*)malloc(durata*sizeof(float));
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
storage=(float*)malloc(N*sizeof(float));

hipMalloc((float**)&sistema.x,N*sizeof(float));
hipMalloc((float**)&sistema.eta,N*sizeof(float));
inizializza<<<blocks,threads>>>(sistema.x,sistema.eta);
hiprandState * stato;
hipMalloc((void**)&stato,N*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(stato);
for(t=0;t<durata;t++){	evolvi<<<blocks,threads>>>(stato,sistema.x,sistema.eta);
			if(t>=termalizzazione)hipMemcpy(&traettoria[t-termalizzazione],sistema.eta,sizeof(float),hipMemcpyDeviceToHost);}
hipMemcpy(storage,sistema.eta,N*sizeof(float),hipMemcpyDeviceToHost);

stampa(storage,N);
stampa_file(traettoria,durata-termalizzazione);
free(storage);
hipFree(sistema.x);
hipFree(sistema.eta);
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
//printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}
