#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int threads=1024;
const int blocks=30;
const int particelle=threads*blocks;
const int N=4;
const float size=6.0;


struct vertici{	float x;
		float y;
			};

struct geometria{	vertici*	punti;};

struct stato{		float*	x;
			float*	y;
			float*	eta_x;
			float* 	eta_y;};
void	stampa(stato configurazione){int i;FILE*f;
					f=fopen("output.dat","w");
		for(i=0;i<particelle;i++){fprintf(f,"%f		%f \n",configurazione.x[i],configurazione.y[i]);}
					fclose(f);}




// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
__global__ inline float isLeft( vertici P0, vertici P1, vertici P2 )
{
    return ( (P1.x - P0.x) * (P2.y - P0.y)
            - (P2.x -  P0.x) * (P1.y - P0.y) );
}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
__global__ int wn_PnPoly( vertici P, vertici* V, int n ){
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (V[i].y <= P.y) {          // start y <= P.y
            if (V[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( V[i], V[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (V[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( V[i], V[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}
//************non viene usato in questa simulazione perchè con il winding number all edges that are totally above or totally below P get rejected after only two (2) inequality tests. However, currently popular implementations of the cn algorithm ) use at least three (3) inequality tests for each rejected edge. 

// cn_PnPoly(): crossing number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  0 = outside, 1 = inside
// This code is patterned after [Franklin, 2000]
int	cn_PnPoly( vertici P, vertici* V, int n ){
   					 int    cn = 0;    // the  crossing number counter

   					 // loop through all edges of the polygon
   				 for (int i=0; i<n; i++) {    // edge from V[i]  to V[i+1]
     					if (((V[i].y <= P.y) && (V[i+1].y > P.y))     // an upward crossing
        				|| ((V[i].y > P.y) && (V[i+1].y <=  P.y))) { // a downward crossing
      	      								// compute  the actual edge-ray intersect x-coordinate
            								float vt = (float)(P.y  - V[i].y) / (V[i+1].y - V[i].y);
            								if (P.x <  V[i].x + vt * (V[i+1].x - V[i].x)) // P.x < intersect
               									  ++cn;   // a valid crossing of y=P.y right of P.x
        									}
    							}
   	 return (cn&1);    // 0 if even (out), and 1 if  odd (in)

}

__global__ bool is_present(geometria scheletro,vertici P){int wn=wn_PnPoly(P,scheletro.punti,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



__global__ void	inizializza(stato* configurazione,geometria scheletro){int id=threadIdx.x+ blockIdx.x*blockDim.x;
						vertici temp;bool flag=false;
						while(flag){
							temp.x=-1+size*(float)hiprand()/RAND_MAX;
							temp.y=-1+size*(float)hiprand()/RAND_MAX;

							if (!is_present(scheletro,temp)){flag=true;
										(*configurazione).x[id]=temp.x;
										(*configurazione).y[id]=temp.y;
											}
 									}					 									 															
					
							}




void alloco_vertici(geometria* scheletro){  int i; 
					(*scheletro).punti=(vertici*)malloc((N+1)*sizeof(vertici));
					for(i=0;i<N;i++){
					printf("inserisci ascissa e ordinata del %d° vertice\n",i);
					scanf("%f",&(*scheletro).punti[i].x);scanf("%f",&(*scheletro).punti[i].y);
							}
					(*scheletro).punti[N].x=(*scheletro).punti[0].x;
					(*scheletro).punti[N].y=(*scheletro).punti[0].y;//sto creando una geometria con un vertice in più che coincide con il primo
					}

main(){
geometria scheletro;
//numero di vertici dell'oggetto
alloco_vertici(&scheletro);
geometria scheletro_dev;
hipMalloc((vertici**)&scheletro_dev.punti,(N+1)*sizeof(vertici));
hipMemcpy(scheletro_dev.punti,scheletro.punti,(N+1)*sizeof(vertici),hipMemcpyHostToDevice);

//for(int i=0;i<N;i++) {scheletro.punti[i].x=0;scheletro.punti[i].y=0;}

//hipMemcpy(scheletro.punti,scheletro_dev.punti,(N+1)*sizeof(vertici),hipMemcpyDeviceToHost);
srand(10);
//for(int i=0;i<N;i++) {printf("%f	%f \n",scheletro.punti[i].x,scheletro.punti[i].y);}


printf("finito di allocare\n");
stato configurazione;

hipMalloc((float**)&configurazione.x,particelle*sizeof(float));

hipMalloc((float**)&configurazione.y,particelle*sizeof(float));
//configurazione.y=(float*)malloc(particelle*sizeof(float));

inizializza<<<blocks,threads>>>(&configurazione,scheletro);
/*
vertici a,b,c;
a.x=0.;
a.y=0.;	b.x=1;b.y=1.;c.x=5.5;c.y=5.7;
if(isLeft(a,b,c)>0) printf("è a sinistra\n");
else printf("è a destra\n");*/
stampa(configurazione);
}


