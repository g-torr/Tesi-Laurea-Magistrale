#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int particelle=1000;
//const int N=4;
const float size=6.0;




struct point{	float x;
		float y;
			};

struct geometria{	point*	vertici;};

struct stato{		float*	x;
			float*	y;
			float*	eta_x;
			float* 	eta_y;};
void	stampa(stato configurazione){int i;FILE*f;
					f=fopen("output.dat","w");
		for(i=0;i<particelle;i++){fprintf(f,"%f		%f \n",configurazione.x[i],configurazione.y[i]);}
					fclose(f);}




float segmento_vicino(point * V, int N, point P,point * closest){ //P è il punto di cui viene calcolata la distanza rispetto al segmento di estremi P1-P2 ; t= indice  
	int i=0;float* t;float d;
	 t=(float*)malloc(N*sizeof(float));
	t[0] = ((P.x-V[0].x)*(V[0+1].x-V[0].x)+(P.y-V[0].y)*(V[0+1].y-V[0].y))/
								((V[0+1].x-V[0].x)*(V[0+1].x-V[0].x)+(V[0+1].y-V[0].y)*(V[0+1].y-V[0].y));

	 if(t[0]<0.0){t[0]=0.0;}
 	 if(t[0]>1.0){t[0]=1.0;} 


    (*closest).x = V[0].x+ (V[0+1].x-V[0].x)*t[0]; 
    (*closest).y = V[0].y+ (V[0+1].y-V[0].y)*t[0];  
	d=(P.x- (*closest).x)*(P.x- (*closest).x)+(P.y- (*closest).y)*(P.y- (*closest).y);

	point temp;
	float d_temp;
 	for(i=1;i<N;i++){
    t[i] = ((P.x-V[i].x)*(V[i+1].x-V[i].x)+(P.y-V[i].y)*(V[i+1].y-V[i].y))/
								((V[i+1].x-V[i].x)*(V[i+1].x-V[i].x)+(V[i+1].y-V[i].y)*(V[i+1].y-V[i].y));

    
    if(t[i]<0.0){t[i]=0.0;}
    if(t[i]>1.0){t[i]=1.0;} 


    temp.x = V[i].x+ (V[i+1].x-V[i].x)*t[i]; 
    temp.y =V[i].y+ (V[i+1].y-V[i].y)*t[i];  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ (*closest).x=temp.x; (*closest).y=temp.y;d=d_temp;}
				}
    
    return d; }

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
inline float	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
int
wn_PnPoly( point P, point* V, int n )
{
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (V[i].y <= P.y) {          // start y <= P.y
            if (V[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( V[i], V[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (V[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( V[i], V[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}




bool is_present(geometria scheletro,point P,int N){int wn=wn_PnPoly(P,scheletro.vertici,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



void	inizializza(stato* configurazione,geometria scheletro,int N){int i=0;point temp;		
						while(i<particelle){
							temp.x=-1+size*(float)rand()/RAND_MAX;
							temp.y=-1+size*(float)rand()/RAND_MAX;
							if (!is_present(scheletro,temp,N)){
										(*configurazione).x[i]=temp.x;(*configurazione).y[i]=temp.y;
											i++;}
 										}					 									 															
					
											}





void alloco_punti(geometria* scheletro,int *N){  int i=0; float a,b;
					FILE*f;f=fopen("input","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					*N=i;rewind(f);
					(*scheletro).vertici=(point*)malloc((*N +1)*sizeof(point));
					for(i=0;i<*N;i++){
							fscanf(f,"%f %f",&(*scheletro).vertici[i].x,&(*scheletro).vertici[i].y);
							}
					fclose(f);
					(*scheletro).vertici[*N].x=(*scheletro).vertici[0].x;
					(*scheletro).vertici[*N].y=(*scheletro).vertici[0].y;//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){
int N;
geometria scheletro;
//numero di punti dell'oggetto
alloco_punti(&scheletro,&N);
//crea_body(&scheletro);
srand(10);
int i;
/*
for(i=0;i<N;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,scheletro.lati[i].x0,scheletro.lati[i].y0,scheletro.lati[i].x1,scheletro.lati[i].y1);}*/

stato configurazione;

configurazione.x=(float*)malloc(particelle*sizeof(float));
configurazione.y=(float*)malloc(particelle*sizeof(float));

inizializza(&configurazione,scheletro,N);
point a,b;a.x=4.;a.y=.5;
float d=segmento_vicino(scheletro.vertici,N,a,&b);
printf("il punto (%f,%f) si trova a distanza%f dal punto (%f,%f)\n",a.x,a.y,sqrt(d),b.x,b.y);
stampa(configurazione);
}


