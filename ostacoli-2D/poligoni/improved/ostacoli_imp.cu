#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>


const int particelle=1000;
//const int N=4;
const float size=3.0;
const float half_size=size/2.;
#define tau 0.06 // costante nel processo di O-U
#define D 0.1
#define dt 0.01
#define mobility 1
const int durata=1000;
const float tsalva= durata/10.;

struct point{	float x;
		float y;
			};

struct geometria{	point*	vertici;};

struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

double randn (double mu, double sigma){
  					double U1, U2, W, mult;
  					static double X1, X2;
  					static int call = 0;
 
  					if (call == 1)
    					{
      						call = !call;
      						return (mu + sigma * (double) X2);
   					 }
 
  					do
   					 {
     						 U1 = -1 + ((double) rand () / RAND_MAX) * 2;
      						U2 = -1 + ((double) rand () / RAND_MAX) * 2;
     						 W = pow (U1, 2) + pow (U2, 2);
   					 }
  						while (W >= 1 || W == 0);
 
				mult = sqrt ((-2 * log (W)) / W);
				X1 = U1 * mult;
  				X2 = U2 * mult;
 
  				call = !call;
 
  return (mu + sigma * (double) X1);
}


point segmento_vicino(point * V, int N, point P){ //P è il punto rispetto al quale viene cercato il segmento più vicino,  la funzione restituisce closest, che è il punto sul perimetro del poligono più vicino a P, d è tale distanza minima.
	int i=0;float t;float d; point closest;
	// t=(float*)malloc(N*sizeof(float));
	t = ((P.x-V[0].x)*(V[0+1].x-V[0].x)+(P.y-V[0].y)*(V[0+1].y-V[0].y))/
								((V[0+1].x-V[0].x)*(V[0+1].x-V[0].x)+(V[0+1].y-V[0].y)*(V[0+1].y-V[0].y));

	 if(t<0.0){t=0.0;}
 	 if(t>1.0){t=1.0;} 


    (closest).x = V[0].x+ (V[0+1].x-V[0].x)*t; 
    (closest).y = V[0].y+ (V[0+1].y-V[0].y)*t;  
	d=(P.x- (closest).x)*(P.x- (closest).x)+(P.y- (closest).y)*(P.y- (closest).y);

	point temp;
	float d_temp;
 	for(i=1;i<N;i++){
    t = ((P.x-V[i].x)*(V[i+1].x-V[i].x)+(P.y-V[i].y)*(V[i+1].y-V[i].y))/
								((V[i+1].x-V[i].x)*(V[i+1].x-V[i].x)+(V[i+1].y-V[i].y)*(V[i+1].y-V[i].y));

    
    if(t<0.0){t=0.0;}
    if(t>1.0){t=1.0;} 


    temp.x = V[i].x+ (V[i+1].x-V[i].x)*t; 
    temp.y =V[i].y+ (V[i+1].y-V[i].y)*t;  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ closest.x=temp.x; closest.y=temp.y;d=d_temp;}
				}

    return closest; }

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
inline float	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
int
wn_PnPoly( point P, point* V, int n )
{
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (V[i].y <= P.y) {          // start y <= P.y
            if (V[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( V[i], V[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (V[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( V[i], V[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}




bool is_in(geometria scheletro,point P,int N){int wn=wn_PnPoly(P,scheletro.vertici,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



void	inizializza(configurazione* stato,geometria scheletro,int N){int i=0;point temp;	
						(*stato).eta=(point*)malloc(particelle*sizeof(point));
						(*stato).r  =(point*)malloc(particelle*sizeof(point));
						(*stato).forza=(point*)calloc(particelle,sizeof(point));
						while(i<particelle){
							temp.x=-half_size+size*(float)rand()/RAND_MAX;
							temp.y=-half_size+size*(float)rand()/RAND_MAX;
							if (!is_in(scheletro,temp,N)){
										(*stato).r[i].x=temp.x;
										(*stato).r[i].y=temp.y;
										(*stato).eta[i].x=0;	
										(*stato).eta[i].y=0;
										i++;}
 										}					 		
											}
void	evolvi(configurazione stato,geometria scheletro, int N,geometria ottimizza){	
			for(int i=0;i<particelle;i++){
						stato.eta[i].x=stato.eta[i].x-(1/tau)*stato.eta[i].x*dt+sqrt(D)*randn(0,sqrt(2.))*sqrt(dt)/    tau;      				//	printf("%f\n",stato.eta[i].x);
						stato.eta[i].y=stato.eta[i].y-(1/tau)*stato.eta[i].y*dt+sqrt(D)*randn(0,sqrt(2.))*sqrt(dt)/tau;
						stato.r[i].x=stato.r[i].x+stato.eta[i].x*dt;
						stato.r[i].y=stato.r[i].y+stato.eta[i].y*dt;	
				if((stato.r[i].x>ottimizza.vertici[0].x)&&(stato.r[i].x<ottimizza.vertici[1].x)&&(stato.r[i].y<ottimizza.vertici[2].x)&&(stato.r[i].y>ottimizza.vertici[0].y)) {		
					if (is_in(scheletro,stato.r[i],N)){
										stato.forza[i]=stato.r[i];
										stato.r[i]=segmento_vicino(scheletro.vertici,N,stato.r[i]);
										stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
										stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);
													}
					else {stato.forza[i].x=0.; stato.forza[i].y=0.;}

}
		else {	if(stato.r[i].x>half_size)	stato.r[i].x=stato.r[i].x-size;
				else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;
			if	(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
				else if (stato.r[i].y<-half_size)	stato.r[i].y=stato.r[i].y+size;
			stato.forza[i].x=0.; stato.forza[i].y=0.;	}
/*					

					if (stato.r[i].x>ottimizza[0].x){
						if(stato.r[i].x<ottimizza[1].x){
							if(stato.r[i].y>ottimizza[0].y){
								if(stato.r[i].y<ottimizza[2].y{
									if (is_in(scheletro,stato.r[i],N)){
										stato.forza[i]=stato.r[i];
										stato.r[i]=segmento_vicino(scheletro.vertici,N,stato.r[i]);
										stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
										stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);
													}
									else	{stato.forza[i].x=0.; stato.forza[i].y=0;}
												}
								else if(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
									stato.forza[i].x=0.; stato.forza[i].y=0;
											}
							else if	(stato.r[i].y<-half_size) stato.r[i].y=stato.r[i].y+size;
									stato.forza[i].x=0.; stato.forza[i].y=0; 											}
					else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;	
						 						}}
				
					else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;
					if	(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
					else if (stato.r[i].y<-half_size)	stato.r[i].y=stato.r[i].y+size;
						if (is_in(scheletro,stato.r[i],N)){ //printf("la particella %d è caduta dentro \n",i);
							stato.forza[i]=stato.r[i];
							stato.r[i]=segmento_vicino(scheletro.vertici,N,stato.r[i]);
							stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
							stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);
}

						else {stato.forza[i].x=0.; stato.forza[i].y=0;}*/
						}
}

void ottimizza_geometria(geometria scheletro, geometria* ottimizza, int N){int i;
					(*ottimizza).vertici=(point*)malloc((4+1)*sizeof(point));
					float min_x,min_y,max_x,max_y;
					(*ottimizza).vertici[0].x= scheletro.vertici[0].x;
					(*ottimizza).vertici[0].y= scheletro.vertici[0].y;
					(*ottimizza).vertici[1].x= scheletro.vertici[0].x;
					(*ottimizza).vertici[2].y= scheletro.vertici[0].y;
					for(i=1;i<N;i++){
						min_x=scheletro.vertici[i].x;
						min_y=scheletro.vertici[i].y;
						max_x=scheletro.vertici[i].x;
						max_y=scheletro.vertici[i].y;
						if ((*ottimizza).vertici[0].x>min_x) (*ottimizza).vertici[0].x=min_x;
						if ((*ottimizza).vertici[1].x<max_x) (*ottimizza).vertici[1].x=max_x; 						
						if ((*ottimizza).vertici[0].y>min_y) (*ottimizza).vertici[0].y=min_y;
						else if ((*ottimizza).vertici[2].y<max_y) (*ottimizza).vertici[2].y=max_y;}
						
						(*ottimizza).vertici[1].y=(*ottimizza).vertici[0].y;
						(*ottimizza).vertici[2].x=(*ottimizza).vertici[1].x;
						(*ottimizza).vertici[3].x=(*ottimizza).vertici[0].x;
						(*ottimizza).vertici[3].y=(*ottimizza).vertici[2].y;
						(*ottimizza).vertici[4].x=(*ottimizza).vertici[0].x;
						(*ottimizza).vertici[4].y=(*ottimizza).vertici[0].y;

}
void alloco_punti(geometria* scheletro,int *N){  int i=0; float a,b;
					FILE*f;f=fopen("input2.dat","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					*N=i;rewind(f);
					(*scheletro).vertici=(point*)malloc((*N +1)*sizeof(point));
					for(i=0;i<*N;i++){
							fscanf(f,"%f %f",&(*scheletro).vertici[i].x,&(*scheletro).vertici[i].y);
							}
					fclose(f);
					(*scheletro).vertici[*N].x=(*scheletro).vertici[0].x;
					(*scheletro).vertici[*N].y=(*scheletro).vertici[0].y;//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){
clock_t t1 = clock();
int N;
geometria scheletro;
geometria ottimizza;
alloco_punti(&scheletro,&N);
ottimizza_geometria(scheletro,&ottimizza,N);
srand(10);
int t,i=0;
/*
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,ottimizza.vertici[i].x,ottimizza.vertici[i].y,ottimizza.vertici[i+1].x,ottimizza.vertici[i+1].y);}*/
configurazione stato;
inizializza(&stato,scheletro,N);
int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);
for(t=0;t<numero_passi;t++){
			evolvi(stato,scheletro,N,ottimizza);
			   if((t% passi_salvataggio==0)&&(t>0)){stampa(stato,i);i++;}
			}
point a;a.x=.25;a.y=-1.5;
point b=segmento_vicino(scheletro.vertici,N,a);
point c;c.x=a.x-b.x; c.y=a.y-b.y;

printf("il punto (%f,%f) si trova a distanza%f dal punto (%f,%f)\n",a.x,a.y,sqrt(c.x*c.x+c.y*c.y),b.x,b.y);
//stampa(stato,1);
 clock_t t2 = clock();
double time_sec = 
       (double)(t2-t1)/(double)(CLOCKS_PER_SEC); 
 
    printf("Time (sec): %lf\n",time_sec); 
}


