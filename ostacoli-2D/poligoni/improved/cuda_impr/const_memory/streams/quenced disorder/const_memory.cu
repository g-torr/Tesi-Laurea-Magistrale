#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
/*#include <hip/hip_runtime_api.h>
#include <helper_functions.h>*/  
const float size=3.0;
const float half_size=size/2.;
#define tau 0.06 // costante nel processo di O-U
#define D 1.
#define dt 0.01
#define mobility 1
#define N 16
const int durata=size*10*sqrt(D/tau);
const float tsalva= durata/10.;
const int blocks=100;
const int threads=1024;
const int particelle=blocks*threads;
struct point{	float x;
		float y;};


struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
__constant__ point vertice[N+1];
__constant__ point ottimizza[4+1];


void creo_cartelle()					{system( "rm -rf ./posizione" );system( "rm -rf ./forza" );
							mkdir("posizione",0700);mkdir("forza",0700);}
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che si utilizza richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}


__device__ void segmento_vicino(point P,point* closest,int id){ //P è il punto rispetto al quale viene cercato il segmento più vicino,  la funzione restituisce closest, che è il punto sul perimetro del poligono più vicino a P, d è tale distanza minima.
	int i=0;float t;float d; 
	 
	t = ((P.x-vertice[0].x)*(vertice[0+1].x-vertice[0].x)+(P.y-vertice[0].y)*(vertice[0+1].y-vertice[0].y))/
								((vertice[0+1].x-vertice[0].x)*(vertice[0+1].x-vertice[0].x)+(vertice[0+1].y-vertice[0].y)*(vertice[0+1].y-vertice[0].y));

	 if(t<0.0){t=0.0;}
 	 if(t>1.0){t=1.0;} 


    closest[id].x = vertice[0].x+ (vertice[0+1].x-vertice[0].x)*t; 
    closest[id].y = vertice[0].y+ (vertice[0+1].y-vertice[0].y)*t;  
	d=(P.x- closest[id].x)*(P.x- closest[id].x)+(P.y- closest[id].y)*(P.y- closest[id].y);

	point temp;
	float d_temp;
 	for(i=1;i<N;i++){
    t = ((P.x-vertice[i].x)*(vertice[i+1].x-vertice[i].x)+(P.y-vertice[i].y)*(vertice[i+1].y-vertice[i].y))/
								((vertice[i+1].x-vertice[i].x)*(vertice[i+1].x-vertice[i].x)+(vertice[i+1].y-vertice[i].y)*(vertice[i+1].y-vertice[i].y));

    
    if(t<0.0){t=0.0;}
    if(t>1.0){t=1.0;} 

	
    temp.x = vertice[i].x+ (vertice[i+1].x-vertice[i].x)*t; 
    temp.y =vertice[i].y+ (vertice[i+1].y-vertice[i].y)*t;  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ closest[id].x=temp.x; closest[id].y=temp.y;d=d_temp;}
				}

     }

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
__device__ inline float	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
__device__ int wn_PnPoly( point P, int n )
{
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (vertice[i].y <= P.y) {          // start y <= P.y
            if (vertice[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( vertice[i], vertice[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (vertice[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( vertice[i], vertice[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}




__device__ bool is_in(point P){int wn=wn_PnPoly(P,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



__global__ void	inizializza(configurazione stato,hiprandState* gen_random){point temp;
							int id=threadIdx.x+blockIdx.x*blockDim.x;	
							float v=sqrt(D/tau);
							do {
							temp.x=-half_size+size*hiprand_uniform(&gen_random[id]);
							temp.y=-half_size+size*hiprand_uniform(&gen_random[id]);}
							
								while(!is_in(temp)); 		 	     							stato.r[id].x=temp.x;
						stato.r[id].y=temp.y;
						stato.eta[id].x=v*hiprand_normal(&gen_random[id]);	
						stato.eta[id].y=v*hiprand_normal(&gen_random[id]);
										
 															 		
}
__global__ void	evolvi(configurazione stato,hiprandState* gen_random){	
						 
						int id=threadIdx.x+blockIdx.x*blockDim.x; 	
						//stato.eta[id].x=stato.eta[id].x-(1/tau)*stato.eta[id].x*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;      			
						//stato.eta[id].y=stato.eta[id].y-(1/tau)*stato.eta[id].y*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;			
						stato.r[id].x=stato.r[id].x+stato.eta[id].x*dt;
						stato.r[id].y=stato.r[id].y+stato.eta[id].y*dt;	
			//	if((stato.r[id].x>ottimizza[0].x)&&(stato.r[id].x<ottimizza[1].x)&&(stato.r[id].y<ottimizza[2].x)&&(stato.r[id].y>ottimizza[0].y)) {		
					if (!is_in(stato.r[id])){
										stato.forza[id]=stato.r[id];
										segmento_vicino(stato.r[id],stato.r,id);
										
										stato.forza[id].x=(stato.forza[id].x-stato.r[id].x)/(dt*mobility);
										stato.forza[id].y=(stato.forza[id].y-stato.r[id].y)/(dt*mobility); 
													}
					else {stato.forza[id].x=0.; stato.forza[id].y=0.;}

									}
		/*else {	if(stato.r[id].x>half_size)	stato.r[id].x=stato.r[id].x-size;
				else if (stato.r[id].x<-half_size) stato.r[id].x=stato.r[id].x+size;
			if	(stato.r[id].y>half_size)	stato.r[id].y=stato.r[id].y-size;
				else if (stato.r[id].y<-half_size)	stato.r[id].y=stato.r[id].y+size;
			stato.forza[id].x=0.; stato.forza[id].y=0.; 	}

						
//if(id==5)printf(" la particella 5 si trova in %f \n",stato.r[id].x);
}*/

void ottimizza_geometria(point* scheletro, point* temp_ottimizza){int i;
				
					float min_x,min_y,max_x,max_y;
					temp_ottimizza[0].x= scheletro[0].x;
					temp_ottimizza[0].y= scheletro[0].y;
					temp_ottimizza[1].x= scheletro[0].x;
					temp_ottimizza[2].y= scheletro[0].y;
					for(i=1;i<N;i++){
						min_x=scheletro[i].x;
						min_y=scheletro[i].y;
						max_x=scheletro[i].x;
						max_y=scheletro[i].y;
						if (temp_ottimizza[0].x>min_x) temp_ottimizza[0].x=min_x;
						if (temp_ottimizza[1].x<max_x) temp_ottimizza[1].x=max_x; 						
						if (temp_ottimizza[0].y>min_y) temp_ottimizza[0].y=min_y;
						else if (temp_ottimizza[2].y<max_y) temp_ottimizza[2].y=max_y;}
						
						temp_ottimizza[1].y=temp_ottimizza[0].y;
						temp_ottimizza[2].x=temp_ottimizza[1].x;
						temp_ottimizza[3].x=temp_ottimizza[0].x;
						temp_ottimizza[3].y=temp_ottimizza[2].y;
						temp_ottimizza[4].x=temp_ottimizza[0].x;
						temp_ottimizza[4].y=temp_ottimizza[0].y;

}
void alloco_punti(point* scheletro){  int i=0; float a,b;
					FILE*f;f=fopen("input2.dat","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					rewind(f); 
					if(N!=i){printf("il tuo file di input non è consistente con il numero di vertici aspettato"); 
							exit(-1);}	
					
					for(i=0;i<N;i++){
							fscanf(f,"%f %f",&scheletro[i].x,&scheletro[i].y);
							}
					fclose(f);
					scheletro[N].x=scheletro[0].x;
					scheletro[N].y=scheletro[0].y;//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){
int deviceCount;
hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
creo_cartelle();
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

int i,t;
point* temp_scheletro;
point* temp_ottimizza;
temp_scheletro=(point*)malloc((N +1)*sizeof(point));
alloco_punti(temp_scheletro);
temp_ottimizza=(point*)malloc((4+1)*sizeof(point));
ottimizza_geometria(temp_scheletro,temp_ottimizza);

printf("per usare la shared memory il numero di vertici  deve essere fissato al tempo di compilazione, sicuro di avere %d vertici?\n",N);
hipMemcpyToSymbol(HIP_SYMBOL(ottimizza),temp_ottimizza,(4+1)*sizeof(point));
hipMemcpyToSymbol(HIP_SYMBOL(vertice),temp_scheletro,(N+1)*sizeof(point));

int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);


/*
for(i=0;i<4;i++){
temp_ottimizza[i].x=0;
temp_ottimizza[i].y=0;}
srand(10);
hipMemcpyFromSymbol(temp_ottimizza,HIP_SYMBOL(ottimizza),(4+1)*sizeof(point));
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,temp_ottimizza[i].x,temp_ottimizza[i].y,temp_ottimizza[i+1].x,temp_ottimizza[i+1].y);}
*/


hipStream_t stream0,stream1,stream2,stream3;
hipStreamCreate(&stream0);hipStreamCreate(&stream1);hipStreamCreate(&stream2);hipStreamCreate(&stream3);//****creo gli stream
configurazione dev_stato_0,dev_stato_1,dev_stato_2,dev_stato_3,stato;//alloco lo stato del sistema sul device e sull'host tramite hipHostAlloc (pinned memory)

hipMalloc((point**)&dev_stato_0.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.forza,(particelle/4)*sizeof(point));

hipMalloc((point**)&dev_stato_1.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_2.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_3.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.forza,(particelle/4)*sizeof(point));

hipHostAlloc((point**)&stato.eta,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.r,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.forza,particelle*sizeof(point),hipHostMallocDefault);


hiprandState * generatori_random;//alloco il generatore dei numeri random sul device
if (numero_passi> pow(2,67)) printf("ATTENZIONE! ricontrollare il generatore di numeri casuali"); 
hipMalloc((void**)&generatori_random,particelle*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(generatori_random);



inizializza<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
inizializza<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
inizializza<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
inizializza<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);

i=0;
for(t=0;t<numero_passi;t++){
//for(t=0;t<10;t++){
			evolvi<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
			evolvi<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
			evolvi<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
			evolvi<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);
	
	   if((t% passi_salvataggio==0)&&(t>0)){printf("siamo a %d/9 \n",i);
			hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
			hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
			hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
			hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
			hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
			hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
			hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
			hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
	  		hipDeviceSynchronize();
			stampa(stato,i);i++;}

			}
 

hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
 
hipDeviceSynchronize();
/*
hipMemcpy(stato.r,dev_stato.r,particelle*sizeof(point),hipMemcpyDeviceToHost);
hipMemcpy(stato.forza,dev_stato.forza,particelle*sizeof(point),hipMemcpyDeviceToHost);*/
stampa(stato,i);
hipFree(dev_stato_0.r);
hipFree(dev_stato_0.eta);
hipFree(dev_stato_0.forza);
hipFree(dev_stato_1.r);
hipFree(dev_stato_1.eta);
hipFree(dev_stato_1.forza);
hipFree(dev_stato_2.r);
hipFree(dev_stato_2.eta);
hipFree(dev_stato_2.forza);
hipFree(dev_stato_3.r);
hipFree(dev_stato_3.eta);
hipFree(dev_stato_3.forza);

hipHostFree(stato.r);
hipHostFree(stato.forza);
hipHostFree(stato.eta);
hipStreamSynchronize(stream0);		
hipStreamSynchronize(stream1);
hipStreamSynchronize(stream2);		
hipStreamSynchronize(stream3);
 hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
hipStreamDestroy(stream0);hipStreamDestroy(stream1);hipStreamDestroy(stream2);hipStreamDestroy(stream3);
}

