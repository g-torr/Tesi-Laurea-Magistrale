#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
#include <ftw.h>
#include <unistd.h>
/*#include <hip/hip_runtime_api.h>
#include <helper_functions.h>*/  
const float size=3.0;
const float half_size=size/2.;
#define tau 0.06 // costante nel processo di O-U
#define D 0.1
#define dt 0.01
#define mobility 1
#define N 4
const int durata=size*size*2/D;
const float tsalva= durata/10.;
const int blocks=100;
const int threads=1024;
const int particelle=blocks*threads;
struct point{	float x;
		float y;};


struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
__constant__ point vertice[N+1];
__constant__ point ottimizza[4+1];

void creo_cartelle()					{system( "rm -rf ./posizione" );system( "rm -rf ./forza" );
							mkdir("posizione",0700);mkdir("forza",0700);}
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che si utilizza richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}

//				non la uso più!!
__device__ void segmento_vicino(point P,point* closest,int id){ //P è il punto rispetto al quale viene cercato il segmento più vicino,  la funzione restituisce closest, che è il punto sul perimetro del poligono più vicino a P, d è tale distanza minima.
	int i=0;float t;float d; 
	 
	t = ((P.x-vertice[0].x)*(vertice[0+1].x-vertice[0].x)+(P.y-vertice[0].y)*(vertice[0+1].y-vertice[0].y))/
								((vertice[0+1].x-vertice[0].x)*(vertice[0+1].x-vertice[0].x)+(vertice[0+1].y-vertice[0].y)*(vertice[0+1].y-vertice[0].y));

	 if(t<0.0){t=0.0;}
 	 if(t>1.0){t=1.0;} 


    closest[id].x = vertice[0].x+ (vertice[0+1].x-vertice[0].x)*t; 
    closest[id].y = vertice[0].y+ (vertice[0+1].y-vertice[0].y)*t;  
	d=(P.x- closest[id].x)*(P.x- closest[id].x)+(P.y- closest[id].y)*(P.y- closest[id].y);

	point temp;
	float d_temp;
 	for(i=1;i<N;i++){
    t = ((P.x-vertice[i].x)*(vertice[i+1].x-vertice[i].x)+(P.y-vertice[i].y)*(vertice[i+1].y-vertice[i].y))/
								((vertice[i+1].x-vertice[i].x)*(vertice[i+1].x-vertice[i].x)+(vertice[i+1].y-vertice[i].y)*(vertice[i+1].y-vertice[i].y));

    
    if(t<0.0){t=0.0;}
    if(t>1.0){t=1.0;} 

	
    temp.x = vertice[i].x+ (vertice[i+1].x-vertice[i].x)*t; 
    temp.y =vertice[i].y+ (vertice[i+1].y-vertice[i].y)*t;  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ closest[id].x=temp.x; closest[id].y=temp.y;d=d_temp;}
				}

     }

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
__device__ inline float	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
__device__ int wn_PnPoly( point P, int n )
{
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (vertice[i].y <= P.y) {          // start y <= P.y
            if (vertice[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( vertice[i], vertice[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (vertice[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( vertice[i], vertice[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}


__device__ bool interseca(float*t, float*T,point P_old,point P_nuovo,point V1,point V2){float x1,x2,X1,X2,y1,y2,Y1,Y2;
						X1=P_old.x;	X2=P_nuovo.x;	x1=V1.x;	x2=V2.x;
						Y1=P_old.y;	Y2=P_nuovo.y;	y1=V1.y;	y2=V2.y;
					*T=(x2*(-y1 + Y1) + X1*(y1 - y2) + x1*(-Y1 + y2))/((X1 - X2)*(y1 - y2) - (x1 - x2)*(Y1 - Y2));
					*t=(X2*(y1 - Y1) + x1*(Y1 - Y2) + X1*(-y1 + Y2))/(-((X1 - X2)*(y1 - y2)) + (x1 - x2)*(Y1 - Y2));					
					if((*t>=0.0)&&(*t<=1.0)&&(*T>=0.)&&(*T<=1.)) return true;
     						else return false;	}

__device__ bool closest(point P_old,point P, point* nuovo){float t,temp_t,T,temp_T;int i;bool flag=false;
				T=1;				
				for(i=0;i<N;i++){ 
				if (interseca(&temp_t,&temp_T,P_old,P,vertice[i],vertice[i+1])){flag=true;if (temp_T<T) T=temp_T;}
						}
				
				(*nuovo).x=(P.x-P_old.x)*(T-0.00000001)+P_old.x;
				(*nuovo).y=(P.y-P_old.y)*(T-0.00000001)+P_old.y;
				return flag;	
					} 
__device__ bool is_in(point P){int wn=wn_PnPoly(P,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



__global__ void	inizializza(configurazione stato,hiprandState* gen_random){point temp;
							int id=threadIdx.x+blockIdx.x*blockDim.x;	
						
							do {
							temp.x=-half_size+size*hiprand_uniform(&gen_random[id]);
							temp.y=-half_size+size*hiprand_uniform(&gen_random[id]);}
							
								while(is_in(temp)); 		 	     							stato.r[id].x=temp.x;
						stato.r[id].y=temp.y;
						stato.eta[id].x=0;	
						stato.eta[id].y=0;
										
 															 		
}
__global__ void	evolvi(configurazione stato,hiprandState* gen_random){	
						 
						int id=threadIdx.x+blockIdx.x*blockDim.x; point proposta;	
						stato.eta[id].x=stato.eta[id].x-(1/tau)*stato.eta[id].x*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;      			
						stato.eta[id].y=stato.eta[id].y-(1/tau)*stato.eta[id].y*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;			
						proposta.x=stato.r[id].x+stato.eta[id].x*dt;
						proposta.y=stato.r[id].y+stato.eta[id].y*dt;	
						stato.forza[id]=proposta;						
						bool flag=closest(stato.r[id],proposta,&stato.r[id]);
		if((stato.r[id].x>ottimizza[0].x)&&(stato.r[id].x<ottimizza[1].x)&&(stato.r[id].y<ottimizza[2].x)&&(stato.r[id].y>ottimizza[0].y))
		{ 
					if(flag){
				//	if (is_in(proposta)){segmento_vicino(proposta,&proposta,0); 				
							
						//	stato.r[id]=proposta;
							stato.forza[id].x=(stato.forza[id].x-stato.r[id].x)/(dt*mobility);
							stato.forza[id].y=(stato.forza[id].y-stato.r[id].y)/(dt*mobility); }
					else{stato.forza[id].x=0.; stato.forza[id].y=0.;}
		}	
		else{
			stato.forza[id].x=0.; stato.forza[id].y=0.; 
			if(stato.r[id].x>half_size) 	stato.r[id].x=stato.r[id].x-size;
				else if (stato.r[id].x<-half_size) stato.r[id].x=stato.r[id].x+size;
			if	(stato.r[id].y>half_size)	stato.r[id].y=stato.r[id].y-size;
				else if (stato.r[id].y<-half_size)	stato.r[id].y=stato.r[id].y+size;
					}
}
				
		
void ottimizza_geometria(point* scheletro, point* temp_ottimizza){int i;
				
					float min_x,min_y,max_x,max_y;
					temp_ottimizza[0].x= scheletro[0].x;
					temp_ottimizza[0].y= scheletro[0].y;
					temp_ottimizza[1].x= scheletro[0].x;
					temp_ottimizza[2].y= scheletro[0].y;
					for(i=1;i<N;i++){
						min_x=scheletro[i].x;
						min_y=scheletro[i].y;
						max_x=scheletro[i].x;
						max_y=scheletro[i].y;
						if (temp_ottimizza[0].x>min_x) temp_ottimizza[0].x=min_x;
						if (temp_ottimizza[1].x<max_x) temp_ottimizza[1].x=max_x; 						
						if (temp_ottimizza[0].y>min_y) temp_ottimizza[0].y=min_y;
						else if (temp_ottimizza[2].y<max_y) temp_ottimizza[2].y=max_y;}
						
						temp_ottimizza[1].y=temp_ottimizza[0].y;
						temp_ottimizza[2].x=temp_ottimizza[1].x;
						temp_ottimizza[3].x=temp_ottimizza[0].x;
						temp_ottimizza[3].y=temp_ottimizza[2].y;
						temp_ottimizza[4].x=temp_ottimizza[0].x;
						temp_ottimizza[4].y=temp_ottimizza[0].y;

}
void alloco_punti(point* scheletro){  int i=0; float a,b;
					FILE*f;f=fopen("input2.dat","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					rewind(f); 
					if(N!=i){printf("il tuo file di input non è consistente con il numero di vertici aspettato"); 
							exit(-1);}	
					
					for(i=0;i<N;i++){
							fscanf(f,"%f %f",&scheletro[i].x,&scheletro[i].y);
							}
					fclose(f);
					scheletro[N].x=scheletro[0].x;
					scheletro[N].y=scheletro[0].y;//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){int deviceCount;
hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
creo_cartelle();
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

int i,t;
point* temp_scheletro;
point* temp_ottimizza;
temp_scheletro=(point*)malloc((N +1)*sizeof(point));
alloco_punti(temp_scheletro);
temp_ottimizza=(point*)malloc((4+1)*sizeof(point));
ottimizza_geometria(temp_scheletro,temp_ottimizza);

printf("per usare la constant memory il numero di vertici  deve essere fissato al tempo di compilazione, sicuro di avere %d vertici?\n",N);
hipMemcpyToSymbol(HIP_SYMBOL(ottimizza),temp_ottimizza,(4+1)*sizeof(point));
hipMemcpyToSymbol(HIP_SYMBOL(vertice),temp_scheletro,(N+1)*sizeof(point));

int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);


/*
for(i=0;i<4;i++){
temp_ottimizza[i].x=0;
temp_ottimizza[i].y=0;}
srand(10);
hipMemcpyFromSymbol(temp_ottimizza,HIP_SYMBOL(ottimizza),(4+1)*sizeof(point));
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,temp_ottimizza[i].x,temp_ottimizza[i].y,temp_ottimizza[i+1].x,temp_ottimizza[i+1].y);}
*/

hipStream_t stream0,stream1,stream2,stream3;
hipStreamCreate(&stream0);hipStreamCreate(&stream1);hipStreamCreate(&stream2);hipStreamCreate(&stream3);//****creo gli stream
configurazione dev_stato_0,dev_stato_1,dev_stato_2,dev_stato_3,stato;//alloco lo stato del sistema sul device e sull'host tramite hipHostAlloc (pinned memory)

hipMalloc((point**)&dev_stato_0.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.forza,(particelle/4)*sizeof(point));

hipMalloc((point**)&dev_stato_1.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_2.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_3.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.forza,(particelle/4)*sizeof(point));

hipHostAlloc((point**)&stato.eta,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.r,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.forza,particelle*sizeof(point),hipHostMallocDefault);
/*
stato.eta=(point*)malloc(particelle*sizeof(point));
stato.r  =(point*)malloc(particelle*sizeof(point));
stato.forza=(point*)calloc(particelle,sizeof(point));*/


hiprandState * generatori_random;//alloco il generatore dei numeri random sul device
if (numero_passi> pow(2,67)) printf("ATTENZIONE! ricontrollare il generatore di numeri casuali"); 
hipMalloc((void**)&generatori_random,particelle*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(generatori_random);


inizializza<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
inizializza<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
inizializza<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
inizializza<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);
i=0;
for(t=0;t<numero_passi;t++){
//for(t=0;t<10;t++){
			evolvi<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
			evolvi<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
			evolvi<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
			evolvi<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);
	
	   if((t% passi_salvataggio==0)&&(t>0)){printf("siamo a %d/9 \n",i);
			hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
			hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
			hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
			hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
			hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
			hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
			hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
			hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
	  		hipDeviceSynchronize();
			stampa(stato,i);i++;}

			}

hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
 
hipDeviceSynchronize();
/*
hipMemcpy(stato.r,dev_stato.r,particelle*sizeof(point),hipMemcpyDeviceToHost);
hipMemcpy(stato.forza,dev_stato.forza,particelle*sizeof(point),hipMemcpyDeviceToHost);*/
stampa(stato,i);
hipFree(dev_stato_0.r);
hipFree(dev_stato_0.eta);
hipFree(dev_stato_0.forza);
hipFree(dev_stato_1.r);
hipFree(dev_stato_1.eta);
hipFree(dev_stato_1.forza);
hipFree(dev_stato_2.r);
hipFree(dev_stato_2.eta);
hipFree(dev_stato_2.forza);
hipFree(dev_stato_3.r);
hipFree(dev_stato_3.eta);
hipFree(dev_stato_3.forza);

hipHostFree(stato.r);
hipHostFree(stato.forza);
hipHostFree(stato.eta);
hipStreamSynchronize(stream0);		
hipStreamSynchronize(stream1);
hipStreamSynchronize(stream2);		
hipStreamSynchronize(stream3);
 hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
hipStreamDestroy(stream0);hipStreamDestroy(stream1);hipStreamDestroy(stream2);hipStreamDestroy(stream3);
}


