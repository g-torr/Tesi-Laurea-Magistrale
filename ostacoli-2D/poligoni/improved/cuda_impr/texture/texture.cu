#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
/*#include <hip/hip_runtime_api.h>
#include <helper_functions.h>*/  
const float size=(float)3.;
const float half_size=size/(float)2.;
#define tau (float)0.06 // costante nel processo di O-U
#define D (float)0.1
#define dt (float)0.01
#define mobility (float)1
#define N 100
const int durata=size*size*2/D;
const float tsalva= durata/10;
const int blocks=128;
const int threads=1024;
const int particelle=blocks*threads;
struct point{	float x;
		float y;};


struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
texture <float,1> text_px;
texture <float,1> text_py;
__constant__ point ottimizza[4+1];

void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
						mkdir("posizione",0700);mkdir("forza",0700);
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che si utilizza richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}


__device__ point segmento_vicino(point P){ //P è il punto rispetto al quale viene cercato il segmento più vicino,  la funzione restituisce closest, che è il punto sul perimetro del poligono più vicino a P, d è tale distanza minima.
	int i=0;float t;float d;  point closest;
	point prev,next;
	prev.x=tex1Dfetch(text_px,0);	next.x=tex1Dfetch(text_px,1);
	prev.y=tex1Dfetch(text_py,0);	next.y=tex1Dfetch(text_py,1);
	 
	t = ((P.x-tex1Dfetch(text_px,0))*(next.x-prev.x)+(P.y-prev.y)*(next.y-prev.y))/
								((next.x-prev.x)*(next.x-prev.x)+(next.y-prev.y)*(next.y-prev.y));

	 if(t<0.0){t=(float)0;}
 	 if(t>1.0){t=(float)1;} 


    closest.x = prev.x+ (next.x-prev.x)*t; 
    closest.y = prev.y+ (next.y-prev.y)*t;  
	d=(P.x- closest.x)*(P.x- closest.x)+(P.y- closest.y)*(P.y- closest.y);

	point temp;
	float d_temp;
 	for(i=1;i<N;i++){	
	prev.x=tex1Dfetch(text_px,i);	next.x=tex1Dfetch(text_px,i+1);
	prev.y=tex1Dfetch(text_py,i);	next.y=tex1Dfetch(text_py,i+1);
    t = ((P.x-prev.x)*(next.x-prev.x)+(P.y-prev.y)*(next.y-prev.y))/
								((next.x-prev.x)*(next.x-prev.x)+(next.y-prev.y)*(next.y-prev.y));

    
    if(t<0.0){t=(float)0;}
    if(t>1.0){t=(float)1;} 

	
    temp.x = prev.x+ (next.x-prev.x)*t; 
    temp.y =prev.y+ (next.y-prev.y)*t;  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ closest.x=temp.x; closest.y=temp.y;d=d_temp;}
				}
return closest;
     }

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
__device__ inline float	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
__device__ int wn_PnPoly( point P )
{	point prev,next;
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
	//#pragma unroll	
    for (int i=0; i<N; i++) {   // edge from V[i] to  V[i+1]
	prev.x=tex1Dfetch(text_px,i);	next.x=tex1Dfetch(text_px,i+1);
	prev.y=tex1Dfetch(text_py,i);	next.y=tex1Dfetch(text_py,i+1);
	
        if (prev.y <= P.y) {          // start y <= P.y
            if (next.y  > P.y)      // an upward crossing
                 if (isLeft( prev, next, P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (next.y  <= P.y)     // a downward crossing
                 if (isLeft( prev, next, P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}




__device__ bool is_in(point P){int wn=wn_PnPoly(P);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



__global__ void	inizializza(configurazione stato,hiprandState* gen_random){point temp;
							int id=threadIdx.x+blockIdx.x*blockDim.x;	
						
							do {
							temp.x=-half_size+size*hiprand_uniform(&gen_random[id]);
							temp.y=-half_size+size*hiprand_uniform(&gen_random[id]);}
							
								while(is_in(temp)); 		 	     							stato.r[id].x=temp.x;
						stato.r[id].y=temp.y;
						stato.eta[id].x=0;	
						stato.eta[id].y=0;
										
 															 		
}
__global__ void	evolvi(configurazione stato,hiprandState* gen_random){	
						 point r,v,f;
						int id=threadIdx.x+blockIdx.x*blockDim.x; 	
						v=stato.eta[id]; r=stato.r[id];
						v.x=v.x-(1/tau)*v.x*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;   	
						v.y=v.y-(1/tau)*v.y*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau; 	
						r.x=r.x+v.x*dt;
						r.y=r.y+v.y*dt;	
						f.x=0; f.y=0;
			//	if((r.x>ottimizza[0].x)&&(r.x<ottimizza[1].x)&&(r.y<ottimizza[2].x)&&(r.y>ottimizza[0].y)){		
					bool in=is_in(r);
					if (in){
										f=r;
										r=segmento_vicino(r);
										
										f.x=(f.x-r.x)/(dt*mobility);
										f.y=(f.y-r.y)/(dt*mobility); 
							}
					

		/*														}	
		else {*/	if(r.x>half_size)	r.x=r.x-size;
				else if (r.x<-half_size) r.x=r.x+size;
			if	(r.y>half_size)	r.y=r.y-size;
				else if (r.y<-half_size)	r.y=r.y+size;
		//	}
						stato.eta[id]=v;	stato.forza[id]=f;	stato.r[id]=r;

						
//if(id==5)printf(" la particella 5 si trova in %f \n",stato.r[id].x);
}

void ottimizza_geometria(float* x,float* y, point* temp_ottimizza){int i;
				
					float min_x,min_y,max_x,max_y;
					temp_ottimizza[0].x= x[0];
					temp_ottimizza[0].y= y[0];
					temp_ottimizza[1].x= x[0];
					temp_ottimizza[2].y= y[0];
					for(i=1;i<N;i++){
						min_x=x[i];
						min_y=y[i];
						max_x=x[i];
						max_y=y[i];
						if (temp_ottimizza[0].x>min_x) temp_ottimizza[0].x=min_x;
						if (temp_ottimizza[1].x<max_x) temp_ottimizza[1].x=max_x; 						
						if (temp_ottimizza[0].y>min_y) temp_ottimizza[0].y=min_y;
						else if (temp_ottimizza[2].y<max_y) temp_ottimizza[2].y=max_y;}
						
						temp_ottimizza[1].y=temp_ottimizza[0].y;
						temp_ottimizza[2].x=temp_ottimizza[1].x;
						temp_ottimizza[3].x=temp_ottimizza[0].x;
						temp_ottimizza[3].y=temp_ottimizza[2].y;
						temp_ottimizza[4].x=temp_ottimizza[0].x;
						temp_ottimizza[4].y=temp_ottimizza[0].y;

}
void alloco_punti(float* scheletro_x,float *scheletro_y){  int i=0; float a,b;
					FILE*f;f=fopen("input2.dat","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					rewind(f); 
					if(N!=i){printf("il tuo file di input non è consistente con il numero di vertici aspettato"); 
							exit(-1);}	
					
					for(i=0;i<N;i++){
							fscanf(f,"%f %f",&scheletro_x[i],&scheletro_y[i]);
							}
					fclose(f);
					scheletro_x[N]=scheletro_x[0];
					scheletro_y[N]=scheletro_y[0];//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

int i,t;
float* temp_scheletro_x;float* temp_scheletro_y;float* dev_scheletro_x;float* dev_scheletro_y;
point* temp_ottimizza;
temp_scheletro_x=(float*)malloc((N +1)*sizeof(float));
temp_scheletro_y=(float*)malloc((N +1)*sizeof(float));
alloco_punti(temp_scheletro_x,temp_scheletro_y);
hipMalloc((float**)&dev_scheletro_x,(N+1)*sizeof(float));
hipMalloc((float**)&dev_scheletro_y,(N+1)*sizeof(float));

hipBindTexture(NULL,text_px,dev_scheletro_x,(N+1)*sizeof(float));	
hipBindTexture(NULL,text_py,dev_scheletro_y,(N+1)*sizeof(float));	

hipMemcpy(dev_scheletro_x,temp_scheletro_x,(N+1)*sizeof(float),hipMemcpyHostToDevice);
hipMemcpy(dev_scheletro_y,temp_scheletro_y,(N+1)*sizeof(float),hipMemcpyHostToDevice);

temp_ottimizza=(point*)malloc((4+1)*sizeof(point));
ottimizza_geometria(temp_scheletro_x,temp_scheletro_y,temp_ottimizza);
/*
printf("per usare la shared memory il numero di vertici  deve essere fissato al tempo di compilazione, sicuro di avere %d vertici?\n",N);
hipMemcpyToSymbol(HIP_SYMBOL(ottimizza),temp_ottimizza,(4+1)*sizeof(point));
hipMemcpyToSymbol(HIP_SYMBOL(vertice),temp_scheletro,(N+1)*sizeof(point));*/


int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);


/*
for(i=0;i<4;i++){
temp_ottimizza[i].x=0;
temp_ottimizza[i].y=0;}
srand(10);
hipMemcpyFromSymbol(temp_ottimizza,HIP_SYMBOL(ottimizza),(4+1)*sizeof(point));
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,temp_ottimizza[i].x,temp_ottimizza[i].y,temp_ottimizza[i+1].x,temp_ottimizza[i+1].y);}
*/
configurazione dev_stato,stato;//alloco lo stato del sistema
hipMalloc((point**)&dev_stato.eta,particelle*sizeof(point));
hipMalloc((point**)&dev_stato.r,particelle*sizeof(point));
hipMalloc((point**)&dev_stato.forza,particelle*sizeof(point));

hiprandState * generatori_random;//alloco il generatore dei numeri random sul device
if (numero_passi> pow(2,67)) printf("ATTENZIONE! ricontrollare il generatore di numeri casuali"); 
hipMalloc((void**)&generatori_random,particelle*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(generatori_random);



inizializza<<<blocks,threads>>>(dev_stato,generatori_random);
stato.eta=(point*)malloc(particelle*sizeof(point));
stato.r  =(point*)malloc(particelle*sizeof(point));
stato.forza=(point*)calloc(particelle,sizeof(point));


i=0;
for(t=0;t<numero_passi;t++){
			evolvi<<<blocks,threads>>>(dev_stato,generatori_random);
			   if((t% passi_salvataggio==0)&&(t>0)){printf("siamo a %d/9 \n",i);
			hipMemcpy(stato.r,dev_stato.r,particelle*sizeof(point),hipMemcpyDeviceToHost);
			hipMemcpy(stato.forza,dev_stato.forza,particelle*sizeof(point),hipMemcpyDeviceToHost);
				stampa(stato,i);i++;
	
}
//hipDeviceSynchronize();
			}

 

hipMemcpy(stato.r,dev_stato.r,particelle*sizeof(point),hipMemcpyDeviceToHost);
hipMemcpy(stato.forza,dev_stato.forza,particelle*sizeof(point),hipMemcpyDeviceToHost);
stampa(stato,i);
hipFree(dev_stato.r);
hipFree(dev_stato.eta);
hipFree(dev_stato.forza);
free(stato.r);
free(stato.forza);
free(stato.eta);
 hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}


