#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>

#include <ftw.h>
#include <unistd.h>

const int particelle=100000;
//const int N=4;
const double size=3.0;
const double half_size=size/2.;
#define tau 0.06 // costante nel processo di O-U
#define D 0.1
#define dt 0.01
#define mobility 1
const int durata=size*size*1/(2*D);
const double tsalva= durata/10.;

struct point{	double x;
		double y;
			};

struct geometria{	point*	vertici;};

struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
void creo_cartelle()					{system( "rm -rf ./posizione" );system( "rm -rf ./forza" );
							mkdir("posizione",0700);mkdir("forza",0700);}
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

double randn (double mu, double sigma){
  					double U1, U2, W, mult;
  					static double X1, X2;
  					static int call = 0;
 
  					if (call == 1)
    					{
      						call = !call;
      						return (mu + sigma * (double) X2);
   					 }
 
  					do
   					 {
     						 U1 = -1 + ((double) rand () / RAND_MAX) * 2;
      						U2 = -1 + ((double) rand () / RAND_MAX) * 2;
     						 W = pow (U1, 2) + pow (U2, 2);
   					 }
  						while (W >= 1 || W == 0);
 
				mult = sqrt ((-2 * log (W)) / W);
				X1 = U1 * mult;
  				X2 = U2 * mult;
 
  				call = !call;
 
  return (mu + sigma * (double) X1);
}


/*point segmento_vicino(point * V, int N, point P){ //P è il punto rispetto al quale viene cercato il segmento più vicino,  la funzione restituisce closest, che è il punto sul perimetro del poligono più vicino a P, d è tale distanza minima.
	int i=0;double t;double d; point closest;
	// t=(double*)malloc(N*sizeof(double));
	t = ((P.x-V[0].x)*(V[0+1].x-V[0].x)+(P.y-V[0].y)*(V[0+1].y-V[0].y))/
								((V[0+1].x-V[0].x)*(V[0+1].x-V[0].x)+(V[0+1].y-V[0].y)*(V[0+1].y-V[0].y));

	 if(t<0.0){t=0.0;}
 	 if(t>1.0){t=1.0;} 


    (closest).x = V[0].x+ (V[0+1].x-V[0].x)*t; 
    (closest).y = V[0].y+ (V[0+1].y-V[0].y)*t;  
	d=(P.x- (closest).x)*(P.x- (closest).x)+(P.y- (closest).y)*(P.y- (closest).y);

	point temp;
	double d_temp;
 	for(i=1;i<N;i++){
    t = ((P.x-V[i].x)*(V[i+1].x-V[i].x)+(P.y-V[i].y)*(V[i+1].y-V[i].y))/
								((V[i+1].x-V[i].x)*(V[i+1].x-V[i].x)+(V[i+1].y-V[i].y)*(V[i+1].y-V[i].y));

    
    if(t<0.0){t=0.0;}
    if(t>1.0){t=1.0;} 


    temp.x = V[i].x+ (V[i+1].x-V[i].x)*t; 
    temp.y =V[i].y+ (V[i+1].y-V[i].y)*t;  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ closest.x=temp.x; closest.y=temp.y;d=d_temp;}
				}

    return closest; }*/

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
inline double	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
int
wn_PnPoly( point P, point* V, int n )
{
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (V[i].y <= P.y) {          // start y <= P.y
            if (V[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( V[i], V[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (V[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( V[i], V[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}

double interseca(point P_old,point P_nuovo,point V1,point V2){double x1,x2,X1,X2,y1,y2,Y1,Y2;double t; double T;
						X1=P_old.x;	X2=P_nuovo.x;	x1=V1.x;	x2=V2.x;
						Y1=P_old.y;	Y2=P_nuovo.y;	y1=V1.y;	y2=V2.y;
					T=(x2*(-y1 + Y1) + X1*(y1 - y2) + x1*(-Y1 + y2))/((X1 - X2)*(y1 - y2) - (x1 - x2)*(Y1 - Y2));
					t=(X2*(y1 - Y1) + x1*(Y1 - Y2) + X1*(-y1 + Y2))/(-((X1 - X2)*(y1 - y2)) + (x1 - x2)*(Y1 - Y2));		 
			
					if((t>=0.0)&&(t<=1.0)&&(T>=0.)&&(T<1.)) return T;
     						else return 2;	}

bool closest(point P_old,point* P,geometria scheletro,int N){double t,temp_t,T,temp_T;int i;bool flag=false;
				T=2;				
				for(i=0;i<N;i++){ 
				temp_T=interseca(P_old,*P,scheletro.vertici[i],scheletro.vertici[i+1]);
										if (temp_T<T){flag=true;T=temp_T;}

						}
				if(flag==true){
				(*P).x=((*P).x-P_old.x)*(T-0.000001)+P_old.x;
				(*P).y=((*P).y-P_old.y)*(T-0.000001)+P_old.y;
						}
			/*	else{(*P).x=0;
				(*P).y=;}*/
				return flag;	
					} 


bool is_in(geometria scheletro,point P,int N){int wn=wn_PnPoly(P,scheletro.vertici,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



void	inizializza(configurazione* stato,geometria scheletro,int N){int i=0;point temp;	
						(*stato).eta=(point*)malloc(particelle*sizeof(point));
						(*stato).r  =(point*)malloc(particelle*sizeof(point));
						(*stato).forza=(point*)calloc(particelle,sizeof(point));
						while(i<particelle){
							temp.x=-half_size+size*(double)rand()/RAND_MAX;
							temp.y=-half_size+size*(double)rand()/RAND_MAX;
							if (!is_in(scheletro,temp,N)){
										(*stato).r[i].x=temp.x;
										(*stato).r[i].y=temp.y;
										(*stato).eta[i].x=0;	
										(*stato).eta[i].y=0;
										i++;}
 										}					 		
											}
void	evolvi(configurazione stato,geometria scheletro, int N,geometria ottimizza){	point proposta;
			for(int i=0;i<particelle;i++){  
						stato.eta[i].x=stato.eta[i].x-(1/tau)*stato.eta[i].x*dt+sqrt(D)*randn(0,sqrt(2.))*sqrt(dt)/    tau;      				//	printf("%f\n",stato.eta[i].x);
						stato.eta[i].y=stato.eta[i].y-(1/tau)*stato.eta[i].y*dt+sqrt(D)*randn(0,sqrt(2.))*sqrt(dt)/tau;
						proposta.x=stato.r[i].x+stato.eta[i].x*dt;
						proposta.y=stato.r[i].y+stato.eta[i].y*dt;
				stato.forza[i].x=proposta.x;
				stato.forza[i].y=proposta.y;
				bool flag=closest(stato.r[i],&proposta,scheletro,N);
				stato.r[i]=proposta;	
				if((proposta.x>ottimizza.vertici[0].x)&&(proposta.x<ottimizza.vertici[1].x)&&(proposta.y<ottimizza.vertici[2].x)&&(proposta.y>ottimizza.vertici[0].y)) 	{		
						if (flag){
								stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
								stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);}
													
					else {stato.forza[i].x=0.; stato.forza[i].y=0.;}

						}
		else {if(stato.r[i].x>half_size)	stato.r[i].x=stato.r[i].x-size;
				else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;
			if	(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
				else if (stato.r[i].y<-half_size)	stato.r[i].y=stato.r[i].y+size;
			stato.forza[i].x=0.; stato.forza[i].y=0.;	}
/*					

					if (stato.r[i].x>ottimizza[0].x){
						if(stato.r[i].x<ottimizza[1].x){
							if(stato.r[i].y>ottimizza[0].y){
								if(stato.r[i].y<ottimizza[2].y{
									if (is_in(scheletro,stato.r[i],N)){
										stato.forza[i]=stato.r[i];
										stato.r[i]=segmento_vicino(scheletro.vertici,N,stato.r[i]);
										stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
										stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);
													}
									else	{stato.forza[i].x=0.; stato.forza[i].y=0;}
												}
								else if(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
									stato.forza[i].x=0.; stato.forza[i].y=0;
											}
							else if	(stato.r[i].y<-half_size) stato.r[i].y=stato.r[i].y+size;
									stato.forza[i].x=0.; stato.forza[i].y=0; 											}
					else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;	
						 						}}
				
					else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;
					if	(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
					else if (stato.r[i].y<-half_size)	stato.r[i].y=stato.r[i].y+size;
						if (is_in(scheletro,stato.r[i],N)){ //printf("la particella %d è caduta dentro \n",i);
							stato.forza[i]=stato.r[i];
							stato.r[i]=segmento_vicino(scheletro.vertici,N,stato.r[i]);
							stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
							stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);
}

						else {stato.forza[i].x=0.; stato.forza[i].y=0;}*/
						}
}

void ottimizza_geometria(geometria scheletro, geometria* ottimizza, int N){int i;
					(*ottimizza).vertici=(point*)malloc((4+1)*sizeof(point));
					double min_x,min_y,max_x,max_y;
					(*ottimizza).vertici[0].x= scheletro.vertici[0].x;
					(*ottimizza).vertici[0].y= scheletro.vertici[0].y;
					(*ottimizza).vertici[1].x= scheletro.vertici[0].x;
					(*ottimizza).vertici[2].y= scheletro.vertici[0].y;
					for(i=1;i<N;i++){
						min_x=scheletro.vertici[i].x;
						min_y=scheletro.vertici[i].y;
						max_x=scheletro.vertici[i].x;
						max_y=scheletro.vertici[i].y;
						if ((*ottimizza).vertici[0].x>min_x) (*ottimizza).vertici[0].x=min_x;
						if ((*ottimizza).vertici[1].x<max_x) (*ottimizza).vertici[1].x=max_x; 						
						if ((*ottimizza).vertici[0].y>min_y) (*ottimizza).vertici[0].y=min_y;
						else if ((*ottimizza).vertici[2].y<max_y) (*ottimizza).vertici[2].y=max_y;}
						
						(*ottimizza).vertici[1].y=(*ottimizza).vertici[0].y;
						(*ottimizza).vertici[2].x=(*ottimizza).vertici[1].x;
						(*ottimizza).vertici[3].x=(*ottimizza).vertici[0].x;
						(*ottimizza).vertici[3].y=(*ottimizza).vertici[2].y;
						(*ottimizza).vertici[4].x=(*ottimizza).vertici[0].x;
						(*ottimizza).vertici[4].y=(*ottimizza).vertici[0].y;

}
void alloco_punti(geometria* scheletro,int *N){  int i=0; double a,b;
					FILE*f;f=fopen("input2.dat","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					*N=i;rewind(f);
					(*scheletro).vertici=(point*)malloc((*N +1)*sizeof(point));
					for(i=0;i<*N;i++){
							fscanf(f,"%lf %lf",&(*scheletro).vertici[i].x,&(*scheletro).vertici[i].y);
							}
					fclose(f);
					(*scheletro).vertici[*N].x=(*scheletro).vertici[0].x;
					(*scheletro).vertici[*N].y=(*scheletro).vertici[0].y;//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){
creo_cartelle();
clock_t t1 = clock();
int N;
geometria scheletro;
geometria ottimizza;
alloco_punti(&scheletro,&N);
ottimizza_geometria(scheletro,&ottimizza,N);
srand(10);
int t,i=0;
/*
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,ottimizza.vertici[i].x,ottimizza.vertici[i].y,ottimizza.vertici[i+1].x,ottimizza.vertici[i+1].y);}*/
configurazione stato;
inizializza(&stato,scheletro,N);
int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);
for(t=0;t<numero_passi;t++){
//for(t=0;t<8;t++){
			evolvi(stato,scheletro,N,ottimizza);
			   if((t% passi_salvataggio==0)&&(t>0)){stampa(stato,i);i++;}
			//stampa(stato,t);}
			}
point a,b,c;a.x=.25;a.y=-1.5; b.x=0.25;b.y=0;

c.x=b.x; c.y=b.y;
closest(a,&b,scheletro,N);


printf("tra il punto (%f,%f) e il punto (%f,%f) si trova (%f,%f)\n",a.x,a.y,c.x,c.y,b.x,b.y);
//stampa(stato,0);
 clock_t t2 = clock();
double time_sec = 
       (double)(t2-t1)/(double)(CLOCKS_PER_SEC); 
 
    printf("Time (sec): %lf\n",time_sec); 
}


