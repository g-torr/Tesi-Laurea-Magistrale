#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

const float size=3.;
const float half_size=size/2.;
#define tau 256 // costante nel processo di O-U
#define D 1.
#define dt 0.001
#define mobility 1
const int durata=(int)size*size*400/D;
const float tsalva= durata/20.;
const float raggio=1.; //ricordati che deve essere minore di half_size
const int blocks=256;
const int threads=1024;
const int particelle=blocks*threads;
__constant__ float costanti[2];
struct point{	float x;
		float y;
			};



struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					mkdir("posizione",0700);mkdir("forza",0700);
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
					for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); 
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che si utilizza richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}

__device__ point nuova_posizione(float raggio,point old){point nuovo;float d=sqrt((old.x*old.x)+(old.y*old.y));
						nuovo.x=raggio*old.x/d;
						nuovo.y=raggio*old.y/d;
						
						return nuovo;}







__device__ bool is_in(float raggio,point P){float d=sqrt(P.x*P.x+P.y*P.y);
							if (d<raggio) return true;
							else return false;
						}



__global__ void	inizializza(configurazione stato,hiprandState* gen_random){point temp; int id=threadIdx.x+blockIdx.x*blockDim.x;	
								do {
							temp.x=-half_size+size*hiprand_uniform(&gen_random[id]);
							temp.y=-half_size+size*hiprand_uniform(&gen_random[id]);}	
							
								while (!is_in(raggio,temp));
						stato.r[id].x=temp.x;
						stato.r[id].y=temp.y;
						stato.eta[id].x=0;	
						stato.eta[id].y=0;
									 		 		
}
__global__ void	evolvi(configurazione stato,hiprandState* gen_random){	
							 point r,v,f;
						int id=threadIdx.x+blockIdx.x*blockDim.x; 	
						/*v.x=__ldg(&stato.eta[id].x);	v.y=__ldg(&stato.eta[id].y);
						r.x=__ldg(&stato.r[id].x);	r.x=__ldg(&stato.r[id].x);*/
						v=stato.eta[id];	r=stato.r[id];/*
						v.x=v.x-(1.f/tau)*v.x*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;   
						v.y=v.y-(1.f/tau)*v.y*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau; 	*/
						v.x=v.x-v.x*costanti[0]+hiprand_normal(&gen_random[id])*costanti[1];   	
						v.y=v.y-v.y*costanti[0]+hiprand_normal(&gen_random[id])*costanti[1]; 	
						r.x=r.x+v.x*dt;
						r.y=r.y+v.y*dt;	
						f.x=0.f; f.y=0.f;
						
					bool in=is_in(raggio,r);
					if (!in){
										f=r;
										r=nuova_posizione(raggio,r);
										
										f.x=(f.x-r.x)/(dt*mobility);
										f.y=(f.y-r.y)/(dt*mobility); 
							}
					

																	
			if(r.x>half_size)	r.x=r.x-size;
				else if (r.x<-half_size) r.x=r.x+size;
			if	(r.y>half_size)	r.y=r.y-size;
				else if (r.y<-half_size)	r.y=r.y+size;
			
						stato.eta[id]=v;	stato.forza[id]=f;	stato.r[id]=r;

						
//if(id==5)printf(" la particella 5 si trova in %f \n",stato.r[id].x);
}


main(){
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);
float costanti_host[2];
costanti_host[0]=dt/tau;
costanti_host[1]=(float)sqrt(2*D* dt)/tau;
hipMemcpyToSymbol(HIP_SYMBOL(costanti),costanti_host,2*sizeof(float));
if(raggio>half_size){printf("attenzione!! raggio deve essere maggiore di half size");return 1;}

hipStream_t stream0,stream1,stream2,stream3;
hipStreamCreate(&stream0);hipStreamCreate(&stream1);hipStreamCreate(&stream2);hipStreamCreate(&stream3);//****creo gli stream
configurazione dev_stato_0,dev_stato_1,dev_stato_2,dev_stato_3,stato;//alloco lo stato del sistema sul device e sull'host tramite hipHostAlloc (pinned memory)

hipMalloc((point**)&dev_stato_0.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.forza,(particelle/4)*sizeof(point));

hipMalloc((point**)&dev_stato_1.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_2.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_3.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.forza,(particelle/4)*sizeof(point));

hipHostAlloc((point**)&stato.eta,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.r,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.forza,particelle*sizeof(point),hipHostMallocDefault);

hiprandState * generatori_random;//alloco il generatore dei numeri random sul device
if (numero_passi> pow(2,67)) printf("ATTENZIONE! ricontrollare il generatore di numeri casuali"); 
hipMalloc((void**)&generatori_random,particelle*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(generatori_random);

inizializza<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
inizializza<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
inizializza<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
inizializza<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);

int i=0;
for(int t=0;t<numero_passi;t++){
//for(t=0;t<10;t++){
			evolvi<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
			evolvi<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
			evolvi<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
			evolvi<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);
	
	   if((t% passi_salvataggio==0)&&(t>0)&&(t>10*passi_salvataggio)){printf("siamo a %d/9 \n",i);
			hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
			hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
			hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
			hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
			hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
			hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
			hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
			hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
	  		hipDeviceSynchronize();
			stampa(stato,i);i++;}

			}
 

hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
 
hipDeviceSynchronize();
stampa(stato,i);
hipFree(dev_stato_0.r);
hipFree(dev_stato_0.eta);
hipFree(dev_stato_0.forza);
hipFree(dev_stato_1.r);
hipFree(dev_stato_1.eta);
hipFree(dev_stato_1.forza);
hipFree(dev_stato_2.r);
hipFree(dev_stato_2.eta);
hipFree(dev_stato_2.forza);
hipFree(dev_stato_3.r);
hipFree(dev_stato_3.eta);
hipFree(dev_stato_3.forza);

hipHostFree(stato.r);
hipHostFree(stato.forza);
hipHostFree(stato.eta);
hipStreamSynchronize(stream0);		
hipStreamSynchronize(stream1);
hipStreamSynchronize(stream2);		
hipStreamSynchronize(stream3);
 hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
hipStreamDestroy(stream0);hipStreamDestroy(stream1);hipStreamDestroy(stream2);hipStreamDestroy(stream3);
}



