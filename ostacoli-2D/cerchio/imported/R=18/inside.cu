#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

const float size=40.;
const float half_size=size/2.;

#define tau (float) 1. // costante nel processo di O-U
#define D (float)1.
#define dt  (float)0.01
#define mobility 1
const int durata=(int)size*size*10/D;
const float tsalva= durata/10.;
const float raggio=18.; //ricordati che deve essere minore di half_size
const int blocks=512;
const int threads=1024;
const int particelle=blocks*threads;

struct point{	float x;
		float y;
			};



struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					mkdir("posizione",0700);mkdir("forza",0700);
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
					for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); 
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che si utilizza richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}

__device__ point nuova_posizione(float raggio,point old){point nuovo;float d=sqrt((old.x*old.x)+(old.y*old.y));
						nuovo.x=raggio*old.x/d;
						nuovo.y=raggio*old.y/d;
						
						return nuovo;}







__device__ bool is_in(float raggio,point P){float d=sqrt(P.x*P.x+P.y*P.y);
							if (d<raggio) return true;
							else return false;
						}



__global__ void	inizializza(configurazione stato,hiprandState* gen_random){point temp; int id=threadIdx.x+blockIdx.x*blockDim.x;	
								do {
							temp.x=-half_size+size*hiprand_uniform(&gen_random[id]);
							temp.y=-half_size+size*hiprand_uniform(&gen_random[id]);}	
							
								while (!is_in(raggio,temp));
						stato.r[id].x=temp.x;
						stato.r[id].y=temp.y;
						stato.eta[id].x=0;	
						stato.eta[id].y=0;
									 		 		
}
__global__ void	evolvi(configurazione stato,hiprandState* gen_random){	
							int id=threadIdx.x+blockIdx.x*blockDim.x;
						stato.eta[id].x=stato.eta[id].x-(1/tau)*stato.eta[id].x*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;      			
						stato.eta[id].y=stato.eta[id].y-(1/tau)*stato.eta[id].y*dt+sqrt(D)*hiprand_normal(&gen_random[id])*sqrt(2.)*sqrt(dt)/tau;			
						stato.r[id].x=stato.r[id].x+stato.eta[id].x*dt;
						stato.r[id].y=stato.r[id].y+stato.eta[id].y*dt;		
					if (!is_in(raggio,stato.r[id])){
										stato.forza[id]=stato.r[id];
										stato.r[id]=nuova_posizione(raggio,stato.r[id]);
										stato.forza[id].x=(stato.forza[id].x-stato.r[id].x)/(dt*mobility);
										stato.forza[id].y=(stato.forza[id].y-stato.r[id].y)/(dt*mobility);
													}


					else {	if(stato.r[id].x>half_size)	stato.r[id].x=stato.r[id].x-size;
							else if (stato.r[id].x<-half_size) stato.r[id].x=stato.r[id].x+size;
						if	(stato.r[id].y>half_size)	stato.r[id].y=stato.r[id].y-size;
							else if (stato.r[id].y<-half_size)	stato.r[id].y=stato.r[id].y+size;
						stato.forza[id].x=0.; stato.forza[id].y=0.;
						}

//printf(" la particella 5 si trova in %f \n",stato.r[id].x);
					
}


main(){
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);
int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);

if(raggio>half_size){printf("attenzione!! raggio deve essere maggiore di half size");return 1;}

hipStream_t stream0,stream1,stream2,stream3;
hipStreamCreate(&stream0);hipStreamCreate(&stream1);hipStreamCreate(&stream2);hipStreamCreate(&stream3);//****creo gli stream
configurazione dev_stato_0,dev_stato_1,dev_stato_2,dev_stato_3,stato;//alloco lo stato del sistema sul device e sull'host tramite hipHostAlloc (pinned memory)

hipMalloc((point**)&dev_stato_0.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_0.forza,(particelle/4)*sizeof(point));

hipMalloc((point**)&dev_stato_1.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_1.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_2.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_2.forza,(particelle/4)*sizeof(point));


hipMalloc((point**)&dev_stato_3.eta,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.r,(particelle/4)*sizeof(point));
hipMalloc((point**)&dev_stato_3.forza,(particelle/4)*sizeof(point));

hipHostAlloc((point**)&stato.eta,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.r,particelle*sizeof(point),hipHostMallocDefault);
hipHostAlloc((point**)&stato.forza,particelle*sizeof(point),hipHostMallocDefault);

hiprandState * generatori_random;//alloco il generatore dei numeri random sul device
if (numero_passi> pow(2,67)) printf("ATTENZIONE! ricontrollare il generatore di numeri casuali"); 
hipMalloc((void**)&generatori_random,particelle*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(generatori_random);

inizializza<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
inizializza<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
inizializza<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
inizializza<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);

int i=0;
for(int t=0;t<numero_passi;t++){
//for(t=0;t<10;t++){
			evolvi<<<blocks/4,threads,0,stream0>>>(dev_stato_0,generatori_random);
			evolvi<<<blocks/4,threads,0,stream1>>>(dev_stato_1,generatori_random);
			evolvi<<<blocks/4,threads,0,stream2>>>(dev_stato_2,generatori_random);
			evolvi<<<blocks/4,threads,0,stream3>>>(dev_stato_3,generatori_random);
	
	   if((t% passi_salvataggio==0)&&(t>0)){printf("siamo a %d/9 \n",i);
			hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
			hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
			hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
			hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
			hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
			hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
			hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
			hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
	  		hipDeviceSynchronize();
			stampa(stato,i);i++;}

			}
 

hipMemcpyAsync(stato.r,dev_stato_0.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0);
hipMemcpyAsync(stato.r + (particelle/4),dev_stato_1.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream1);
hipMemcpyAsync(stato.r + (particelle/2),dev_stato_2.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream2);
hipMemcpyAsync(stato.r + (3*particelle/4),dev_stato_3.r,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream3);
	
			
hipMemcpyAsync(stato.forza,dev_stato_0.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost,stream0); 			
hipMemcpyAsync(stato.forza + (particelle/4),dev_stato_1.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream1);
hipMemcpyAsync(stato.forza + (particelle/2),dev_stato_2.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream2);
hipMemcpyAsync(stato.forza + (3*particelle/4),dev_stato_3.forza,(particelle/4)*sizeof(point),hipMemcpyDeviceToHost, stream3);
 
hipDeviceSynchronize();
stampa(stato,i);
hipFree(dev_stato_0.r);
hipFree(dev_stato_0.eta);
hipFree(dev_stato_0.forza);
hipFree(dev_stato_1.r);
hipFree(dev_stato_1.eta);
hipFree(dev_stato_1.forza);
hipFree(dev_stato_2.r);
hipFree(dev_stato_2.eta);
hipFree(dev_stato_2.forza);
hipFree(dev_stato_3.r);
hipFree(dev_stato_3.eta);
hipFree(dev_stato_3.forza);

hipHostFree(stato.r);
hipHostFree(stato.forza);
hipHostFree(stato.eta);
hipStreamSynchronize(stream0);		
hipStreamSynchronize(stream1);
hipStreamSynchronize(stream2);		
hipStreamSynchronize(stream3);
 hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
hipStreamDestroy(stream0);hipStreamDestroy(stream1);hipStreamDestroy(stream2);hipStreamDestroy(stream3);
}



