#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

const int particelle=10000;
//const int N=4;
const float size=3.0;
const float half_size=size/2.;
#define tau 0.06 // costante nel processo di O-U
#define D 0.1
#define dt 0.01
#define mobility 1
const int durata=100;
const float tsalva= durata/10.;
const float raggio=1.; //ricordati che deve essere minore di half_size

struct point{	float x;
		float y;
			};



struct configurazione{	point*	eta;
			point*	r;
			point*	forza;};
void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					mkdir("posizione",0700);mkdir("forza",0700);
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}
point nuova_posizione(float raggio,point old){point nuovo;float d=sqrt((old.x*old.x)+(old.y*old.y));
						nuovo.x=raggio*old.x/d;
						nuovo.y=raggio*old.y/d;
						
						return nuovo;}

double randn (double mu, double sigma){
  					double U1, U2, W, mult;
  					static double X1, X2;
  					static int call = 0;
 
  					if (call == 1)
    					{
      						call = !call;
      						return (mu + sigma * (double) X2);
   					 }
 
  					do
   					 {
     						 U1 = -1 + ((double) rand () / RAND_MAX) * 2;
      						U2 = -1 + ((double) rand () / RAND_MAX) * 2;
     						 W = pow (U1, 2) + pow (U2, 2);
   					 }
  						while (W >= 1 || W == 0);
 
				mult = sqrt ((-2 * log (W)) / W);
				X1 = U1 * mult;
  				X2 = U2 * mult;
 
  				call = !call;
 
  return (mu + sigma * (double) X1);
}





bool is_in(float raggio,point P){float d=sqrt(P.x*P.x+P.y*P.y);
							if (d<raggio) return true;
							else return false;
						}



void	inizializza(configurazione* stato,float raggio){int i=0;point temp;	
						(*stato).eta=(point*)malloc(particelle*sizeof(point));
						(*stato).r  =(point*)malloc(particelle*sizeof(point));
						(*stato).forza=(point*)calloc(particelle,sizeof(point));
						while(i<particelle){
							temp.x=-half_size+size*(float)rand()/RAND_MAX;
							temp.y=-half_size+size*(float)rand()/RAND_MAX;
							if (!is_in(raggio,temp)){
										(*stato).r[i].x=temp.x;
										(*stato).r[i].y=temp.y;
										(*stato).eta[i].x=0;	
										(*stato).eta[i].y=0;
										i++;}
 										}					 		
											}
void	evolvi(configurazione stato,float raggio){	
			for(int i=0;i<particelle;i++){
						stato.eta[i].x=stato.eta[i].x-(1/tau)*stato.eta[i].x*dt+sqrt(D)*randn(0,sqrt(2.))*sqrt(dt)/    tau;      				//	printf("%f\n",stato.eta[i].x);
						stato.eta[i].y=stato.eta[i].y-(1/tau)*stato.eta[i].y*dt+sqrt(D)*randn(0,sqrt(2.))*sqrt(dt)/tau;
						stato.r[i].x=stato.r[i].x+stato.eta[i].x*dt;
						stato.r[i].y=stato.r[i].y+stato.eta[i].y*dt;		
					if (is_in(raggio,stato.r[i])){
										stato.forza[i]=stato.r[i];
										stato.r[i]=nuova_posizione(raggio,stato.r[i]);
										stato.forza[i].x=(stato.forza[i].x-stato.r[i].x)/(dt*mobility);
										stato.forza[i].y=(stato.forza[i].y-stato.r[i].y)/(dt*mobility);
													}


					else {	if(stato.r[i].x>half_size)	stato.r[i].x=stato.r[i].x-size;
							else if (stato.r[i].x<-half_size) stato.r[i].x=stato.r[i].x+size;
						if	(stato.r[i].y>half_size)	stato.r[i].y=stato.r[i].y-size;
							else if (stato.r[i].y<-half_size)	stato.r[i].y=stato.r[i].y+size;
						stato.forza[i].x=0.; stato.forza[i].y=0.;
						}

				}	
}


main(){
clock_t t1 = clock();
if(raggio>half_size){printf("attenzione!! raggio deve essere maggiore di half size");return 1;}


srand(10);
int t,i=0;
/*
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,ottimizza.vertici[i].x,ottimizza.vertici[i].y,ottimizza.vertici[i+1].x,ottimizza.vertici[i+1].y);}*/
configurazione stato;
inizializza(&stato,raggio);
int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);
for(t=0;t<numero_passi;t++){
			evolvi(stato,raggio);
			   if((t% passi_salvataggio==0)&&(t>0)){stampa(stato,i);i++;}
			}
point a;a.x=.25;a.y=-1.5;
point b=nuova_posizione(raggio,a);
point c;c.x=a.x-b.x; c.y=a.y-b.y;

printf("il punto (%f,%f) si trova a distanza%f dal punto (%f,%f)\n",a.x,a.y,sqrt(c.x*c.x+c.y*c.y),b.x,b.y);
//stampa(stato,1);
 clock_t t2 = clock();
double time_sec = 
       (double)(t2-t1)/(double)(CLOCKS_PER_SEC); 
 
    printf("Time (sec): %lf\n",time_sec); 
}


