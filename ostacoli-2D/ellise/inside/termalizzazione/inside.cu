#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <hip/hip_runtime_api.h>
/*#include <hip/hip_runtime_api.h>
#include <helper_functions.h>*/  
const float size=(float)3.;
const float half_size=size/(float)2.;
#define tau (float)0.06 // costante nel processo di O-U
#define D (float)0.1
#define dt (float)0.01
#define mobility (float)1
#define N 100
const int durata=size*size*2/D;
const float tsalva= durata/10;
const int blocks=128;
const int threads=1024;
const int particelle=blocks*threads;
struct point{	float x;
		float y;};


struct configurazione{	point*	eta;
			point*	r;
			point*	forza;
			bool* 	inside;};
__constant__ point vertice[N+1];
__constant__ point ottimizza[4+1];
__constant__ float costanti[2];

void creo_cartelle()					{system( "rm -rf ./posizione" );system( "rm -rf ./forza" );
							mkdir("posizione",0700);mkdir("forza",0700);}

void	stampa(configurazione stato, int i){int id;FILE*f;FILE*g;char indirizzo_posizione[50];char indirizzo_forza[50];
					sprintf(indirizzo_posizione,"./posizione/dati_%d",i);
					sprintf(indirizzo_forza,"./forza/forza_%d",i);
	
					f=fopen(indirizzo_posizione,"w");g=fopen(indirizzo_forza,"w");
		for(id=0;id<particelle;id++){fprintf(f,"%f		%f \n",stato.r[id].x,stato.r[id].y); //printf("sto scrivendo %d\n",id);
					fprintf(g,"%f		%f \n",stato.forza[id].x,stato.forza[id].y);}
					fclose(f);fclose(g);}

__global__ void setup_random_kernel(hiprandState* stato){
			int id=threadIdx.x+ blockIdx.x*blockDim.x; 
			hiprand_init(1234,id,0,&stato[id]); //inizializzo lo stato che mi genera i numeri casuali hiprand_init (seed,sequence,offset, hiprandState_t *state). per evitare ogni tipo di problemi qui si è scelto lo stesso seme per tutti i thread, ogni thread avrà sequenza differente: ovvero partendo dallo stesso seme si scartano i primi id*2^67 numeri casuali) a meno che il successivo  codice che si utilizza richiede 2^67 numeri casuali, noi siamo tranquilli che non ci siano overlap di numeri casuali. Si potrebbe cambiare il seme per ogni indice, fissare la sequenza a 0: si  potrebbero avere problemi di correlazione tra le sequenze di numeri con semi diversi(cosa molto rara). Dato che effettivamente l'idea di sprecare 2*67 numeri casuali  per ogni thread mi sembra una follia, forse la soluzione più semplice potrebbe essere quella di giocare sull'offset, in effetti se si fissano seme e sequence, mentre si  fissa il parametro offset= k*id, con k un qualsiasi numero > #numeri casuali che uso in ogni  tread  dovrei essere tranquillo che non si abbiano sovrapposizioni
			}


__device__ point segmento_vicino(point P){ //P è il punto rispetto al quale viene cercato il segmento più vicino,  la funzione restituisce closest, che è il punto sul perimetro del poligono più vicino a P, d è tale distanza minima.
	int i=0;float t;float d; 
	 point closest;
	t = ((P.x-vertice[0].x)*(vertice[1].x-vertice[0].x)+(P.y-vertice[0].y)*(vertice[0+1].y-vertice[0].y))/
								((vertice[0+1].x-vertice[0].x)*(vertice[0+1].x-vertice[0].x)+(vertice[0+1].y-vertice[0].y)*(vertice[0+1].y-vertice[0].y));

	 if(t<(float)0.0){t=(float)0.0;}
 	 if(t>(float)1.0){t=(float)1.0;} 


    closest.x = vertice[0].x+ (vertice[0+1].x-vertice[0].x)*t; 
    closest.y = vertice[0].y+ (vertice[0+1].y-vertice[0].y)*t;  
	d=pow(P.x- closest.x,2)+pow(P.y- closest.y,2);

	point temp;
	float d_temp;
 	for(i=1;i<N;i++){
    t = ((P.x-vertice[i].x)*(vertice[i+1].x-vertice[i].x)+(P.y-vertice[i].y)*(vertice[i+1].y-vertice[i].y))/
								((vertice[i+1].x-vertice[i].x)*(vertice[i+1].x-vertice[i].x)+(vertice[i+1].y-vertice[i].y)*(vertice[i+1].y-vertice[i].y));

    
    if(t<(float)0.0){t=(float)0.0;}
    if(t>(float)1.0){t=(float)1.0;} 

	
    temp.x = vertice[i].x+ (vertice[i+1].x-vertice[i].x)*t; 
    temp.y =vertice[i].y+ (vertice[i+1].y-vertice[i].y)*t;  
	d_temp=(P.x-temp.x)*(P.x-temp.x)+(P.y-temp.y)*(P.y-temp.y);
			if(d_temp<d){ closest.x=temp.x; closest.y=temp.y;d=d_temp;}
			}

	return closest;
     }

// isLeft(): tests if a point is Left|On|Right of an infinite line.
//    Input:  three points P0, P1, and P2
//    Return: >0 for P2 left of the line through P0 and P1
//            =0 for P2  on the line
//            <0 for P2  right of the line
//    See: Algorithm 1 "Area of Triangles and Polygons"
__device__ inline float	isLeft( point P0, point P1, point P2 ){
    							return ( (P1.x - P0.x) * (P2.y - P0.y)- (P2.x -  P0.x) * (P1.y - P0.y) );
				}


// wn_PnPoly(): winding number test for a point in a polygon
//      Input:   P = a point,
//               V[] = vertex points of a polygon V[n+1] with V[n]=V[0]
//      Return:  wn = the winding number (=0 only when P is outside)
__device__ int wn_PnPoly( point P, int n )
{
    int    wn = 0;    // the  winding number counter

    // loop through all edges of the polygon
    for (int i=0; i<n; i++) {   // edge from V[i] to  V[i+1]
        if (vertice[i].y <= P.y) {          // start y <= P.y
            if (vertice[i+1].y  > P.y)      // an upward crossing
                 if (isLeft( vertice[i], vertice[i+1], P) > 0)  // P left of  edge
                     ++wn;            // have  a valid up intersect
        }
        else {                        // start y > P.y (no test needed)
            if (vertice[i+1].y  <= P.y)     // a downward crossing
                 if (isLeft( vertice[i], vertice[i+1], P) < 0)  // P right of  edge
                     --wn;            // have  a valid down intersect
        }
    }
    return wn;
}




__device__ bool is_in(point P){int wn=wn_PnPoly(P,N);
					//printf("il wn=%d \n",wn);
					if (wn>0) return true;
							else return false;
						}



__global__ void	inizializza(configurazione stato,hiprandState* gen_random){point temp;
							int id=threadIdx.x+blockIdx.x*blockDim.x;	
						
							do {
							temp.x=-half_size+size*hiprand_uniform(&gen_random[id]);
							temp.y=-half_size+size*hiprand_uniform(&gen_random[id]);}
							
								while(is_in(temp)); 		 	     							stato.r[id].x=temp.x;
						stato.r[id].y=temp.y;
						stato.eta[id].x=0.f;	
						stato.eta[id].y=0.f;
										
 															 		
}
__global__ void	evolvi(configurazione stato,hiprandState* gen_random){	
			point r,v,f;
			int id=threadIdx.x+blockIdx.x*blockDim.x; 	
			v=stato.eta[id];	r=stato.r[id];
			v.x=v.x-v.x*costanti[0]+hiprand_normal(&gen_random[id])*costanti[1];   	
			v.y=v.y-v.y*costanti[0]+hiprand_normal(&gen_random[id])*costanti[1]; 	
			r.x=r.x+v.x*dt;
			r.y=r.y+v.y*dt;	
			//if((r.x>ottimizza[0].x)&&(r.x<ottimizza[1].x)&&(r.y<ottimizza[2].x)&&(r.y>ottimizza[0].y)){		
					bool in=is_in(r);
					if(!in) stato.inside[id]=true;
					else {stato.inside[id]=false;f.x=0.f; f.y=0.f;stato.forza[id]=f;	
						/*if(r.x>half_size)	r.x=r.x-size;
						else if (r.x<-half_size) r.x=r.x+size;
						if	(r.y>half_size)	r.y=r.y-size;
						else if (r.y<-half_size)	r.y=r.y+size;*/}
												
			stato.eta[id]=v;		stato.r[id]=r;		
				/*	if (in){
										f=r;
										r=segmento_vicino(r);
										
										f.x=(f.x-r.x)/(dt*mobility);
										f.y=(f.y-r.y)/(dt*mobility); 
							}
					

																}	
		else {	if(r.x>half_size)	r.x=r.x-size;
				else if (r.x<-half_size) r.x=r.x+size;
			if	(r.y>half_size)	r.y=r.y-size;
				else if (r.y<-half_size)	r.y=r.y+size;
			}
						stato.eta[id]=v;	stato.forza[id]=f;	stato.r[id]=r;

						*/
}

__global__ void correct(configurazione stato,hiprandState* gen_random,int * ids,int count){
				point r,v,f;
				int k=threadIdx.x+blockIdx.x*blockDim.x;
				while(k<count){
				
				int id=ids[k]; 	
				v=stato.eta[id];	r=stato.r[id];
				f=r;
				r=segmento_vicino(r);
				f.x=(f.x-r.x)/(dt*mobility);
				f.y=(f.y-r.y)/(dt*mobility); 
				stato.eta[id]=v;	stato.forza[id]=f;	stato.r[id]=r;		
				k=k+(gridDim.x*blockDim.x);}
					}

void ottimizza_geometria(point* scheletro, point* temp_ottimizza){int i;
				
					float min_x,min_y,max_x,max_y;
					temp_ottimizza[0].x= scheletro[0].x;
					temp_ottimizza[0].y= scheletro[0].y;
					temp_ottimizza[1].x= scheletro[0].x;
					temp_ottimizza[2].y= scheletro[0].y;
					for(i=1;i<N;i++){
						min_x=scheletro[i].x;
						min_y=scheletro[i].y;
						max_x=scheletro[i].x;
						max_y=scheletro[i].y;
						if (temp_ottimizza[0].x>min_x) temp_ottimizza[0].x=min_x;
						if (temp_ottimizza[1].x<max_x) temp_ottimizza[1].x=max_x; 						
						if (temp_ottimizza[0].y>min_y) temp_ottimizza[0].y=min_y;
						else if (temp_ottimizza[2].y<max_y) temp_ottimizza[2].y=max_y;}
						
						temp_ottimizza[1].y=temp_ottimizza[0].y;
						temp_ottimizza[2].x=temp_ottimizza[1].x;
						temp_ottimizza[3].x=temp_ottimizza[0].x;
						temp_ottimizza[3].y=temp_ottimizza[2].y;
						temp_ottimizza[4].x=temp_ottimizza[0].x;
						temp_ottimizza[4].y=temp_ottimizza[0].y;

}
void alloco_punti(point* scheletro){  int i=0; float a,b;
					FILE*f;f=fopen("input.dat","r");
					while(fscanf(f,"%f" "%f",&a,&b)>0)i++;
					rewind(f); 
					if(N!=i){printf("il tuo file di input non è consistente con il numero di vertici aspettato"); 
							exit(-1);}	
					
					for(i=0;i<N;i++){
							fscanf(f,"%f %f",&scheletro[i].x,&scheletro[i].y);
							}
					fclose(f);
					scheletro[N].x=scheletro[0].x;
					scheletro[N].y=scheletro[0].y;//sto creando una geometria con un vertice in più che coincide con il primo		
					}

main(){
creo_cartelle();
hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

int i,t;
point* temp_scheletro;
point* temp_ottimizza;
temp_scheletro=(point*)malloc((N +1)*sizeof(point));
alloco_punti(temp_scheletro);
temp_ottimizza=(point*)malloc((4+1)*sizeof(point));
ottimizza_geometria(temp_scheletro,temp_ottimizza);

printf("per usare la constant memory il numero di vertici  deve essere fissato al tempo di compilazione, sicuro di avere %d vertici?\n",N);
hipMemcpyToSymbol(HIP_SYMBOL(ottimizza),temp_ottimizza,(4+1)*sizeof(point));
hipMemcpyToSymbol(HIP_SYMBOL(vertice),temp_scheletro,(N+1)*sizeof(point));

float costanti_host[2];
costanti_host[0]=dt/tau;
costanti_host[1]=(float)sqrt(2*D* dt)/tau;
hipMemcpyToSymbol(HIP_SYMBOL(costanti),costanti_host,2*sizeof(float));
int numero_passi=(int)durata/dt;
int passi_salvataggio =(int)(tsalva/dt);


/*
for(i=0;i<4;i++){
temp_ottimizza[i].x=0;
temp_ottimizza[i].y=0;}
srand(10);
hipMemcpyFromSymbol(temp_ottimizza,HIP_SYMBOL(ottimizza),(4+1)*sizeof(point));
for(i=0;i<4;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,temp_ottimizza[i].x,temp_ottimizza[i].y,temp_ottimizza[i+1].x,temp_ottimizza[i+1].y);}
*/
configurazione dev_stato,stato;//alloco lo stato del sistema
hipMalloc((point**)&dev_stato.eta,particelle*sizeof(point));
hipMalloc((point**)&dev_stato.r,particelle*sizeof(point));
hipMalloc((point**)&dev_stato.forza,particelle*sizeof(point));
hipMalloc((bool**)&dev_stato.inside,particelle*sizeof(bool));

hiprandState * generatori_random;//alloco il generatore dei numeri random sul device
if (numero_passi> pow(2,67)) printf("ATTENZIONE! ricontrollare il generatore di numeri casuali"); 
hipMalloc((void**)&generatori_random,particelle*sizeof(hiprandState));
setup_random_kernel<<<blocks,threads>>>(generatori_random);



inizializza<<<blocks,threads>>>(dev_stato,generatori_random);
stato.eta=(point*)malloc(particelle*sizeof(point));
stato.r  =(point*)malloc(particelle*sizeof(point));
stato.forza=(point*)calloc(particelle,sizeof(point));
stato.inside=(bool*)malloc(particelle*sizeof(bool));

i=0;int count;
int*  ids;int*  dev_ids;
hipMalloc((int**)&dev_ids,particelle*sizeof(int));
ids=(int*)malloc(particelle*sizeof(int));
for(t=0;t<numero_passi;t++){
			evolvi<<<blocks,threads>>>(dev_stato,generatori_random);
			hipMemcpy(stato.inside,dev_stato.inside,particelle*sizeof(bool),hipMemcpyDeviceToHost);		
			count=0;
			for(int j=0;j<particelle;j++){
					if(stato.inside[j]==true){ ids[count]=j;count++;}
							}
		//	printf("numero di pallette che entrano %d \n", count);
			
			hipMemcpy(dev_ids,ids,count*sizeof(int),hipMemcpyHostToDevice);
			correct<<<count/1024,1024>>>(dev_stato,generatori_random,dev_ids,count);					
							
			   if((t> 2**i)&&(t>0)){printf("siamo a %d/9 \n",i);
			hipMemcpy(stato.r,dev_stato.r,particelle*sizeof(point),hipMemcpyDeviceToHost);
			hipMemcpy(stato.forza,dev_stato.forza,particelle*sizeof(point),hipMemcpyDeviceToHost);
				stampa(stato,i);i++;
	
}
//hipDeviceSynchronize();
			}

 

hipMemcpy(stato.r,dev_stato.r,particelle*sizeof(point),hipMemcpyDeviceToHost);
hipMemcpy(stato.forza,dev_stato.forza,particelle*sizeof(point),hipMemcpyDeviceToHost);
stampa(stato,i);
hipFree(dev_stato.r);
hipFree(dev_stato.eta);
hipFree(dev_stato.forza);
free(stato.r);
free(stato.forza);
free(stato.eta);
 hipEventRecord(stop,0);
hipEventSynchronize(stop);
float tempo_trascorso; hipEventElapsedTime(&tempo_trascorso,start,stop);
printf("il tempo necessario per eseguire il programma è %f ms\n",tempo_trascorso);
}


