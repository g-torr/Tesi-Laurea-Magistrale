#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

const int particelle=100;
const int N=4;
const float size=3.0;

struct segmenti	{	float x0;
			float x1;
			float y0;
			float y1;		//poichè il segmento è definito a partire da 2 punti, devo passare 4 coordinate per segmento
			bool up;	// questo parametro deve essere assegnato a mano quando si costruisce la geometria, mi dice se devo 						guardare sopra o sotto rispetto a ogni segmento
		};


struct vertici{	float x;
		float y;
			};

struct geometria{	vertici*	punti;
			segmenti*	lati;};

struct stato{		float*	x;
			float*	y;
			float*	eta_x;
			float* 	eta_y;};
void	stampa(stato configurazione){int i;FILE*f;
					f=fopen("output.dat","w");
		for(i=0;i<particelle;i++){fprintf(f,"%f		%f \n",configurazione.x[i],configurazione.y[i]);}
					fclose(f);}

float  		vincolo(segmenti lato,float x, float y){
						float f=y-lato.y1-(((lato.y1-lato.y0)/(lato.x1-lato.x0))*(x-lato.x1));
							return f;}

bool is_present(geometria scheletro,float x, float y){int i;bool flag;
				for(i=0;i<N;i++){
					if ((vincolo(scheletro.lati[i],x,y)>0)==scheletro.lati[i].up) flag=true;
					else return false;
							}return true;}



void	inizializza(stato* configurazione,geometria scheletro){int i=0;float temp_x,temp_y;		
						while(i<particelle){
							temp_x=size*(float)rand()/RAND_MAX;
							temp_y=size*(float)rand()/RAND_MAX;
							if (is_present(scheletro,temp_x,temp_y)){
										(*configurazione).x[i]=temp_x;(*configurazione).y[i]=temp_y;
											i++;}
 										}					 									 															
					
											}



void	crea_body(geometria* scheletro){ int i;int flag;
						(*scheletro).lati=(segmenti*)malloc(N*sizeof(segmenti));
						for(i=0;i<N-1;i++)	{(*scheletro).lati[i].x0=(*scheletro).punti[i].x;
									(*scheletro).lati[i].y0=(*scheletro).punti[i].y;
									(*scheletro).lati[i].x1=(*scheletro).punti[i+1].x;
									(*scheletro).lati[i].y1=(*scheletro).punti[i+1].y;
									flag=2;	
									while((flag!=1)&&(flag!=-1))	{
									printf("se il vincolo del lato %d è >0 premi 1; altrimenti se è <0 premi -1\n",i);												
									scanf("%d",&flag); }
									if (flag==1)	(*scheletro).lati[i].up=true;
									else if (flag==-1) (*scheletro).lati[i].up=false;
 									}
						(*scheletro).lati[N-1].x0=(*scheletro).punti[N-1].x;
						(*scheletro).lati[N-1].y0=(*scheletro).punti[N-1].y;
						(*scheletro).lati[N-1].x1=(*scheletro).punti[0].x;
						(*scheletro).lati[N-1].y1=(*scheletro).punti[0].y;
					flag=2;	
					while((flag!=1)&&(flag!=-1))	{
					printf("se il vincolo del lato %d è >0 premi 1; altrimenti se è <0 premi -1\n",i);
					scanf("%d",&flag);		}
					if (flag==1)	(*scheletro).lati[i].up=true;
					else if (flag==-1) (*scheletro).lati[i].up=false;	}


void alloco_vertici(geometria* scheletro){  int i; 
					(*scheletro).punti=(vertici*)malloc(N*sizeof(vertici));
					for(i=0;i<N;i++){
					printf("inserisci ascissa e ordinata del %d° vertice\n",i);
					scanf("%f",&(*scheletro).punti[i].x);scanf("%f",&(*scheletro).punti[i].y);
							}
						}

main(){
geometria scheletro;
//numero di vertici dell'oggetto
alloco_vertici(&scheletro);
crea_body(&scheletro);
srand(10);
int i;

for(i=0;i<N;i++){
					printf("gli estremi del segmento %d° vertice sono (%f,%f) 	(%f,%f)\n",i,scheletro.lati[i].x0,scheletro.lati[i].y0,scheletro.lati[i].x1,scheletro.lati[i].y1);}

stato configurazione;

configurazione.x=(float*)malloc(particelle*sizeof(float));
configurazione.y=(float*)malloc(particelle*sizeof(float));

inizializza(&configurazione,scheletro);

stampa(configurazione);
}


